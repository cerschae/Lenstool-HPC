#include "hip/hip_runtime.h"
#include <fstream>
#include "grid_gradient_GPU.cuh"

#define BLOCK_SIZE 16

void
module_potentialDerivatives_totalGradient_SOA_CPU_GPU(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);

void
module_potentialDerivatives_totalGradient_SOA_CPU_GPU_v2(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);

void calculate_cossin_values(double *theta_cos, double *theta_sin, double *angles, int nhalos ){
	for(int i = 0 ; i < nhalos; i++){
		theta_cos[i]=cos(angles[i]);
		theta_sin[i]=sin(angles[i]);
	}
}

void gradient_grid_GPU_sorted(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells){


  int nBlocks_gpu = 0;
  // Define the number of threads per block the GPU will use
  hipDeviceProp_t properties_gpu;

  hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

  if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
    {
    fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
    exit(-1);
    }
  else
    {
    nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
                    // per Block that the GPU supports
    }

  grid_param *frame_gpu;
  Potential_SOA *lens_gpu,*lens_kernel ;
  int *type_gpu;
  double *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
  double *grid_grad_x_gpu, *grid_grad_y_gpu ;

  lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
  lens_gpu->type = (int *) malloc(sizeof(int));

  // Allocate variables on the GPU
  cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradientgpu.cu : Alloc Potential_SOA: " );
  cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradientgpu.cu : Alloc type_gpu: " );
  cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc x_gpu: " );
  cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc y_gpu: " );
  cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc b0_gpu: " );
  cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc angle_gpu: " );
  cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc epot_gpu: " );
  cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc rcore_gpu: " );
  cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc rcut_gpu: " );
  cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc anglecos_gpu: " );
  cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc anglesin_gpu: " );
  cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradientgpu.cu : Alloc frame_gpu: " );
  cudasafe(hipMalloc( (void**)&(grid_grad_x_gpu), (nbgridcells) * (nbgridcells) *sizeof(double)),"Gradientgpu.cu : Alloc source_x_gpu: " );
  cudasafe(hipMalloc( (void**)&(grid_grad_y_gpu), (nbgridcells) * (nbgridcells) *sizeof(double)),"Gradientgpu.cu : Alloc source_y_gpu: " );

  // Copy values to the GPU
  cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy type_gpu: " );
  cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy x_gpu: " );
  cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy y_gpu: " );
  cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy b0_gpu: " );
  cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy angle_gpu: " );
  cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy epot_gpu: " );
  cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy rcore_gpu: " );
  cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy rcut_gpu: " );
  cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy anglecos: " );
  cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy anglesin: " );
  cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy fame_gpu: " );


  //printf("%p \n", lens_gpu);
  //printf("%p \n", type_gpu);
  //printf("%p \n", lens_gpu->type);
  //fflush(stdout);
  lens_gpu->type = type_gpu;
  lens_gpu->position_x = lens_x_gpu;
  lens_gpu->position_y = lens_y_gpu;
  lens_gpu->b0 = b0_gpu;
  lens_gpu->ellipticity_angle = angle_gpu;
  lens_gpu->ellipticity_potential = epot_gpu;
  lens_gpu->rcore = rcore_gpu;
  lens_gpu->rcut = rcut_gpu;
  lens_gpu->anglecos = anglecos_gpu;
  lens_gpu->anglesin = anglesin_gpu;

  hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);


  if (int((nbgridcells) * (nbgridcells)/threadsPerBlock) == 0){
    gradient_grid_kernel<<<1,threadsPerBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu,frame_gpu,nhalos, nbgridcells, lens_kernel);
  }
  else{
    gradient_grid_kernel<<<(nbgridcells) * (nbgridcells)/threadsPerBlock,threadsPerBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu,frame_gpu,nhalos, nbgridcells, lens_kernel);
  }
  cudasafe(hipMemcpy( grid_grad_x, grid_grad_x_gpu, (nbgridcells) * (nbgridcells) *sizeof(double),hipMemcpyDeviceToHost ),"Gradientgpu.cu : Copy source_x_gpu: " );
  cudasafe(hipMemcpy( grid_grad_y, grid_grad_y_gpu, (nbgridcells) * (nbgridcells) *sizeof(double), hipMemcpyDeviceToHost),"Gradientgpu.cu : Copy source_y_gpu: " );


  //printf("%f %f \n",grid_grad_x[0],grid_grad_y[0]);

    // Free GPU memory
  hipFree(lens_gpu);
  hipFree(type_gpu);
  hipFree(lens_x_gpu);
  hipFree(lens_y_gpu);
  hipFree(b0_gpu);
  hipFree(angle_gpu);
  hipFree(epot_gpu);
  hipFree(rcore_gpu);
  hipFree(rcut_gpu);
  hipFree(anglecos_gpu);
  hipFree(anglesin_gpu);
  hipFree(grid_grad_x_gpu);
  hipFree(grid_grad_y_gpu);

/*
  for (int i = 0; i < nbgridcells; i++){
    for(int j = 0; j < nbgridcells; j++){
      printf(" %f",grid_grad_x[i*nbgridcells + j]);
    }
    printf("\n");
  }*/

}

void gradient_grid_GPU_multiple(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int nbgridcells){
  //get number of GPU devices
  int nDevices;
  hipGetDeviceCount(&nDevices);

  // Initialise kernel variables, table for multiple devices
  grid_param *frame_gpu[nDevices];
  Potential_SOA *lens_gpu[nDevices],*lens_kernel[nDevices] ;
  int *type_gpu[nDevices];
  double *lens_x_gpu[nDevices], *lens_y_gpu[nDevices], *b0_gpu[nDevices], *angle_gpu[nDevices], *epot_gpu[nDevices], *rcore_gpu[nDevices], *rcut_gpu[nDevices],*anglecos_gpu[nDevices], *anglesin_gpu[nDevices];
  double *grid_grad_x_gpu[nDevices], *grid_grad_y_gpu[nDevices] ;



  // Initialise multiple device variables
  int Ndevice[nDevices], indexactual[nDevices];
  hipStream_t stream[nDevices];

  indexactual[0] = 0 ;
  Ndevice[0] = (nbgridcells) * (nbgridcells)/nDevices;

  printf("Using %d Gpu's \n",nDevices );
  //printf("%d %d \n",indexactual[0], Ndevice[0]);

  for (int dev = 1; dev < nDevices; dev++) {

    Ndevice[dev] = (nbgridcells) * (nbgridcells)/nDevices;

    if(indexactual[dev]+Ndevice[dev] > (nbgridcells) * (nbgridcells)){
      Ndevice[dev] = (nbgridcells) * (nbgridcells) - indexactual[dev-1];
    }

    indexactual[dev] = indexactual[dev-1] + Ndevice[dev];
    //printf("%d %d \n",indexactual[dev], Ndevice[dev]);
  }

  for (int dev = 0; dev < nDevices; dev++) {


    hipSetDevice(dev);

    lens_gpu[dev] = (Potential_SOA *) malloc(sizeof(Potential_SOA));
    lens_gpu[dev]->type = (int *) malloc(sizeof(int));

    // Allocate variables on the GPU
    cudasafe(hipMalloc( (void**)&(lens_kernel[dev]), sizeof(Potential_SOA)),"Gradientgpu.cu : Alloc Potential_SOA: " );
    cudasafe(hipMalloc( (void**)&(type_gpu[dev]), nhalos*sizeof(int)),"Gradientgpu.cu : Alloc type_gpu: " );
    cudasafe(hipMalloc( (void**)&(lens_x_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc x_gpu: " );
    cudasafe(hipMalloc( (void**)&(lens_y_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc y_gpu: " );
    cudasafe(hipMalloc( (void**)&(b0_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc b0_gpu: " );
    cudasafe(hipMalloc( (void**)&(angle_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc angle_gpu: " );
    cudasafe(hipMalloc( (void**)&(epot_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc epot_gpu: " );
    cudasafe(hipMalloc( (void**)&(rcore_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc rcore_gpu: " );
    cudasafe(hipMalloc( (void**)&(rcut_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc rcut_gpu: " );
    cudasafe(hipMalloc( (void**)&(frame_gpu[dev]), sizeof(grid_param)),"Gradientgpu.cu : Alloc frame_gpu: " );
    cudasafe(hipMalloc( (void**)&(anglecos_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc anglecos_gpu: " );
    cudasafe(hipMalloc( (void**)&(anglesin_gpu[dev]), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc anglesin_gpu: " );
    cudasafe(hipMalloc( (void**)&(grid_grad_x_gpu[dev]), Ndevice[dev] *sizeof(double)),"Gradientgpu.cu : Alloc source_x_gpu: " );
    cudasafe(hipMalloc( (void**)&(grid_grad_y_gpu[dev]), Ndevice[dev] *sizeof(double)),"Gradientgpu.cu : Alloc source_y_gpu: " );

    hipStreamCreate(&stream[dev]);

  }

  for (int dev = 0; dev < nDevices; dev++) {


    hipSetDevice(dev);

    // Copy values to the GPU
    cudasafe(hipMemcpyAsync(type_gpu[dev],lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice,stream[dev] ),"Gradientgpu.cu : Copy type_gpu: " );
    cudasafe(hipMemcpyAsync(lens_x_gpu[dev],lens->position_x , nhalos*sizeof(double),hipMemcpyHostToDevice,stream[dev] ),"Gradientgpu.cu : Copy x_gpu: " );
    cudasafe(hipMemcpyAsync(lens_y_gpu[dev],lens->position_y , nhalos*sizeof(double), hipMemcpyHostToDevice,stream[dev]),"Gradientgpu.cu : Copy y_gpu: " );
    cudasafe(hipMemcpyAsync(b0_gpu[dev],lens->b0 , nhalos*sizeof(double), hipMemcpyHostToDevice,stream[dev]),"Gradientgpu.cu : Copy b0_gpu: " );
    cudasafe(hipMemcpyAsync(angle_gpu[dev],lens->ellipticity_angle , nhalos*sizeof(double), hipMemcpyHostToDevice,stream[dev]),"Gradientgpu.cu : Copy angle_gpu: " );
    cudasafe(hipMemcpyAsync(epot_gpu[dev], lens->ellipticity_potential, nhalos*sizeof(double),hipMemcpyHostToDevice ,stream[dev]),"Gradientgpu.cu : Copy epot_gpu: " );
    cudasafe(hipMemcpyAsync(rcore_gpu[dev], lens->rcore, nhalos*sizeof(double),hipMemcpyHostToDevice ,stream[dev]),"Gradientgpu.cu : Copy rcore_gpu: " );
    cudasafe(hipMemcpyAsync(rcut_gpu[dev], lens->rcut, nhalos*sizeof(double), hipMemcpyHostToDevice,stream[dev]),"Gradientgpu.cu : Copy rcut_gpu: " );
    cudasafe(hipMemcpyAsync(anglecos_gpu[dev], lens->anglecos, nhalos*sizeof(double),hipMemcpyHostToDevice,stream[dev] ),"Gradientgpu.cu : Copy anglecos: " );
    cudasafe(hipMemcpyAsync(anglesin_gpu[dev], lens->anglesin, nhalos*sizeof(double), hipMemcpyHostToDevice,stream[dev]),"Gradientgpu.cu : Copy anglesin: " );
    cudasafe(hipMemcpyAsync(frame_gpu[dev], frame, sizeof(grid_param), hipMemcpyHostToDevice,stream[dev]),"Gradientgpu.cu : Copy fame_gpu: " );


    lens_gpu[dev]->type = type_gpu[dev];
    lens_gpu[dev]->position_x = lens_x_gpu[dev];
    lens_gpu[dev]->position_y = lens_y_gpu[dev];
    lens_gpu[dev]->b0 = b0_gpu[dev];
    lens_gpu[dev]->ellipticity_angle = angle_gpu[dev];
    lens_gpu[dev]->ellipticity_potential = epot_gpu[dev];
    lens_gpu[dev]->rcore = rcore_gpu[dev];
    lens_gpu[dev]->rcut = rcut_gpu[dev];
    lens_gpu[dev]->anglecos = anglecos_gpu[dev];
    lens_gpu[dev]->anglesin = anglesin_gpu[dev];

    hipMemcpyAsync(lens_kernel[dev], lens_gpu[dev], sizeof(Potential_SOA), hipMemcpyHostToDevice,stream[dev]);
  }

  for (int dev = 0; dev < nDevices; dev++) {


    hipSetDevice(dev);
    int nBlocks_gpu = 0;
    hipDeviceProp_t properties_gpu;
    hipGetDeviceProperties(&properties_gpu, dev); // Get properties of 0th GPU in use

    if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
      {
      fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
      exit(-1);
      }


    if (int((nbgridcells) * (nbgridcells)/threadsPerBlock) == 0){
    	gradient_grid_kernel_multiple<<<1,threadsPerBlock,0,stream[dev]>>>(grid_grad_x_gpu[dev], grid_grad_y_gpu[dev],frame_gpu[dev],nhalos, nbgridcells, lens_kernel[dev], indexactual[dev],Ndevice[dev]);
    }
    else{
    	gradient_grid_kernel_multiple<<<(nbgridcells) * (nbgridcells)/threadsPerBlock,threadsPerBlock,0,stream[dev]>>>(grid_grad_x_gpu[dev], grid_grad_y_gpu[dev],frame_gpu[dev],nhalos, nbgridcells, lens_kernel[dev], indexactual[dev],Ndevice[dev]);
    }
  }

  for (int dev = 0; dev < nDevices; dev++) {
    cudasafe(hipMemcpyAsync( grid_grad_x + indexactual[dev], grid_grad_x_gpu[dev], Ndevice[dev] *sizeof(double),hipMemcpyDeviceToHost ,stream[dev]),"Gradientgpu.cu : Copy source_x_gpu: " );
    cudasafe(hipMemcpyAsync( grid_grad_y + indexactual[dev], grid_grad_y_gpu[dev], Ndevice[dev] *sizeof(double), hipMemcpyDeviceToHost,stream[dev]),"Gradientgpu.cu : Copy source_y_gpu: " );
  }

  for (int dev = 0; dev < nDevices; dev++) {


    hipSetDevice(dev);
      // Free GPU memory
    hipFree(type_gpu[dev]);
    hipFree(lens_x_gpu[dev]);
    hipFree(lens_y_gpu[dev]);
    hipFree(b0_gpu[dev]);
    hipFree(angle_gpu[dev]);
    hipFree(epot_gpu[dev]);
    hipFree(rcore_gpu[dev]);
    hipFree(rcut_gpu[dev]);
    hipFree(anglecos_gpu[dev]);
    hipFree(anglesin_gpu[dev]);
    hipFree(grid_grad_x_gpu[dev]);
    hipFree(grid_grad_y_gpu[dev]);
    hipStreamDestroy(stream[dev]);

  }

}

#if 1
void gradient_grid_GPU_sub(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int nbgridcells, int indexactual, int Ncells ){


	// GPU Property query
	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;

	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

	if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
	{
		fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
		exit(-1);
	}
	else
	{
		nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
		// per Block that the GPU supports
	}

	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel ;
	int *type_gpu;
	double *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	double *grid_grad_x_gpu, *grid_grad_y_gpu ;

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradientgpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradientgpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(double)),"Gradientgpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradientgpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad_x_gpu), (nbgridcells) * (nbgridcells) *sizeof(double)),"Gradientgpu.cu : Alloc source_x_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad_y_gpu), (nbgridcells) * (nbgridcells) *sizeof(double)),"Gradientgpu.cu : Alloc source_y_gpu: " );

	// Copy values to the GPU
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(double),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(double), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy fame_gpu: " );


	//printf("%p \n", lens_gpu);
	//printf("%p \n", type_gpu);
	//printf("%p \n", lens_gpu->type);
	//fflush(stdout);
	lens_gpu->type = type_gpu;
	lens_gpu->position_x = lens_x_gpu;
	lens_gpu->position_y = lens_y_gpu;
	lens_gpu->b0 = b0_gpu;
	lens_gpu->ellipticity_angle = angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore = rcore_gpu;
	lens_gpu->rcut = rcut_gpu;
	lens_gpu->anglecos = anglecos_gpu;
	lens_gpu->anglesin = anglesin_gpu;

	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
#if 0
	int BLOCK_SIZE = 16; // number of threads
	int GRID_SIZE = (nbgridcells + BLOCK_SIZE - 1)/BLOCK_SIZE; // number of blocks
	//
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(GRID_SIZE, GRID_SIZE);
	//
	if (int((nbgridcells) * (nbgridcells)/threadsPerBlock) == 0)
	{
		gradient_grid_kernel<<<1,threadsPerBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu,frame_gpu,nhalos, nbgridcells, lens_kernel);
	}
	else
	{
		//gradient_grid_kernel<<<(nbgridcells) * (nbgridcells)/threadsPerBlock,threadsPerBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu,frame_gpu,nhalos, nbgridcells, lens_kernel);
		//gradient_grid_kernel_v2<<<dimGrid, dimBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu, frame_gpu, nhalos, nbgridcells, lens_kernel);
		//gradient_grid_kernel_v2<<<dimGrid, dimBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu, frame_gpu, nhalos, nbgridcells, lens_kernel);


	if (int((Ncells)/threadsPerBlock) == 0){
	gradient_grid_kernel_multiple<<<1,threadsPerBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu,frame_gpu,nhalos, nbgridcells, lens_kernel, indexactual, Ncells);
	}
	else{
	gradient_grid_kernel_multiple<<<(Ncells)/threadsPerBlock,threadsPerBlock>>>(grid_grad_x_gpu, grid_grad_y_gpu,frame_gpu,nhalos, nbgridcells, lens_kernel, indexactual, Ncells);
	}
#endif
		module_potentialDerivatives_totalGradient_SOA_CPU_GPU(grid_grad_x_gpu, grid_grad_y_gpu, frame_gpu, lens, lens_kernel, nbgridcells, nhalos);
	cudasafe(hipMemcpy( grid_grad_x, grid_grad_x_gpu, (nbgridcells) * (nbgridcells) *sizeof(double),hipMemcpyDeviceToHost ),"Gradientgpu.cu : Copy source_x_gpu: " );
	cudasafe(hipMemcpy( grid_grad_y, grid_grad_y_gpu, (nbgridcells) * (nbgridcells) *sizeof(double), hipMemcpyDeviceToHost),"Gradientgpu.cu : Copy source_y_gpu: " );


	//printf("%f %f \n",grid_grad_x[0],grid_grad_y[0]);

	// Free GPU memory
	hipFree(lens_gpu);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(grid_grad_x_gpu);
	hipFree(grid_grad_y_gpu);

	/*
	   for (int i = 0; i < nbgridcells; i++){
	   for(int j = 0; j < nbgridcells; j++){
	   printf(" %f",grid_grad_x[i*nbgridcells + j]);
	   }
	   printf("\n");
	   }*/

}


__global__ void gradient_grid_kernel(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, int Nlens,int nbgridcells, const struct Potential_SOA *lens) {

	//*grad_x = *grad_y = 0.;

	int bid=blockIdx.x; // index of the block (and of the set of images)
	int tid=threadIdx.x; // index of the thread within the block

	double dx,dy;        //pixelsize
	int grid_dim, index;
	struct point image_point, Grad;
	dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
	dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
	grid_dim = (nbgridcells);

	index = bid * threadsPerBlock + tid ;

	while(index < grid_dim*grid_dim){

		grid_grad_x[index] = 0.;
		grid_grad_y[index] = 0.;

		image_point.x = frame->xmin + (index/grid_dim)*dx;
		image_point.y = frame->ymin + (index % grid_dim)*dy;

		Grad = module_potentialDerivatives_totalGradient_SOA_GPU(&image_point, lens, Nlens);

		grid_grad_x[index] = Grad.x;
		grid_grad_y[index] = Grad.y;

		bid += gridDim.x;
		index = bid * threadsPerBlock + tid;
	}
}

__global__ void gradient_grid_kernel_v2(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, int Nlens,int nbgridcells, const struct Potential_SOA *lens) {

	//*grad_x = *grad_y = 0.;

	int bid = blockIdx.x; // index of the block (and of the set of images)
	int tid = threadIdx.x; // index of the thread within the block

	double dx,dy;        //pixelsize
	int grid_dim, index;
	struct point image_point, Grad;
	//
	dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
	dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
	//
	grid_dim = (nbgridcells);
	//
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//
	index = col*nbgridcells + row ;
	//
	//while(index < grid_dim*grid_dim){

	//grid_grad_x[index] = 0.;
	//grid_grad_y[index] = 0.;

	image_point.x = frame->xmin + col*dx;
	image_point.y = frame->ymin + row*dy;

	Grad = module_potentialDerivatives_totalGradient_SOA_GPU(&image_point, lens, Nlens);

	grid_grad_x[index] = Grad.x;
	grid_grad_y[index] = Grad.y;

	bid += gridDim.x;
	index = bid * threadsPerBlock + tid;
	//}
}

/*
void gradient_grid_general_CPU(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, int Nlens,int nbgridcells, const struct Potential_SOA *lens){
  int bid=0; // index of the block (and of the set of images)
  int tid=0; // index of the thread within the block

  double dx,dy,x_pos,y_pos;        //pixelsize
  int grid_dim, index;
  point Grad, image_point, true_coord_rotation;
  double      R;
  dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
  dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
  grid_dim = (nbgridcells);

  index = bid ;

  while(index < grid_dim*grid_dim){

    grid_grad_x[index] = 0.;
    grid_grad_y[index] = 0.;

    image_point.x = frame->xmin + (index/grid_dim)*dx;
    image_point.y = frame->ymin + (index % grid_dim)*dy;

    Grad = module_potentialDerivatives_totalGradient_SOA(&image_point, lens, Nlens);

    grid_grad_x[index] = Grad.x;
    grid_grad_y[index] = Grad.y;

    bid += 1;
    index = bid * 1 + tid;
  }
}

*/

__global__
void 
inline
module_potentialDerivatives_totalGradient_8_SOA_GPU(double *grid_grad_x, double *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame, int nbgridcells, int shalos, int nhalos)
{
        //asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
        // 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
        //
        struct point grad, clumpgrad, image_point;
        grad.x = 0;
        grad.y = 0;
	//
	int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
	//
	if ((row < nbgridcells) && (col < nbgridcells))
	{
		//
		int index = col*nbgridcells + row;
		//
		//grid_grad_x[index] = 0.;
		//grid_grad_y[index] = 0.;
		//
		double dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
		double dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
		//
#if 0
		__shared__ double img_pt[2];
		if ((row == 0) && (col == 0))
		{
			img_pt[0] = frame->xmin + col*dx;
			img_pt[1] = frame->ymin + row*dy;
		}
		__syncthreads();
#else
		image_point.x = frame->xmin + col*dx;
		image_point.y = frame->ymin + row*dy;	
#endif
		//
		//
		for(int i = shalos; i < shalos + nhalos; i++)
		{
			//IACA_START;
			//
			struct point true_coord, true_coord_rot; //, result;
			//double       R, angular_deviation;
			complex      zis;
			//
			//result.x = result.y = 0.;
			//
#if 0
			true_coord.x = img_pt[0] - __ldg(&lens->position_x[i]);
			true_coord.y = img_pt[1] - __ldg(&lens->position_y[i]);
#else
			true_coord.x = image_point.x - __ldg(&lens->position_x[i]);
			true_coord.y = image_point.y - __ldg(&lens->position_y[i]);
#endif
			double cosi = __ldg(&lens->anglecos[i]);
			double sinu = __ldg(&lens->anglesin[i]);
			// positionning at the potential center
			// Change the origin of the coordinate system to the center of the clump
			double x = true_coord.x*cosi + true_coord.y*sinu;
			double y = true_coord.y*cosi - true_coord.x*sinu;
			//
			double eps = __ldg(&lens->ellipticity_potential[i]);
			//
			double sqe  = sqrt(eps);
			//
			double rem2 = x*x/((1. + eps)*(1. + eps)) + y*y/((1. - eps)*(1. - eps));
			//
			complex zci;
			complex znum, zden, zres;
			double norm;
			//
			zci.im  = -0.5*(1. - eps*eps)/sqe;
			//
			double rc  = __ldg(&lens->rcore[i]);
			double cx1  = (1. - eps)/(1. + eps);
			znum.re = cx1*x;
			znum.im = 2.*sqe*sqrt(rc*rc + rem2) - y/cx1;
			//
			zden.re = x;
			zden.im = 2.*rc*sqe - y;
			norm    = (zden.re*zden.re + zden.im*zden.im);     // zis = znum/zden
			//
			zis.re  = (znum.re*zden.re + znum.im*zden.im)/norm;
			zis.im  = (znum.im*zden.re - znum.re*zden.im)/norm;
			//
			norm    = zis.re;
			//
			zis.re  = log(sqrt(norm*norm + zis.im*zis.im));  // ln(zis) = ln(|zis|)+i.Arg(zis)
			zis.im  = atan2(zis.im, norm);
			//
			zres.re = zci.im*zis.im;   // Re( zci*ln(zis) )
			zres.im = zci.im*zis.re;   // Im( zci*ln(zis) )
			//
			double b0  = __ldg(&lens->b0[i]);
			grad.x += b0*(zres.re*cosi - zres.im*sinu);
			grad.y += b0*(zres.im*cosi + zres.re*sinu);
		}
		//IACA_END;
		//
		grid_grad_x[index] += grad.x;
		grid_grad_y[index] += grad.y;
	}
}

__global__
void
module_potentialDerivatives_totalGradient_8_SOA_GPU_v2(double *grid_grad_x, double *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame,
 int nbgridcells, int i, int nhalos)
{
        //asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
        // 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
        //
        struct point grad, clumpgrad, image_point;
        grad.x = 0;
        grad.y = 0;
        //
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        //
        if ((row < nbgridcells) && (col < nbgridcells))
        {
                //
                int index = col*nbgridcells + row;
                //
                //grid_grad_x[index] = 0.;
                //grid_grad_y[index] = 0.;
                //
                double dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
                double dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
                //
#if 0
                __shared__ double img_pt[2];
                if ((row == 0) && (col == 0))
                {
                        img_pt[0] = frame->xmin + col*dx;
                        img_pt[1] = frame->ymin + row*dy;
                }
                __syncthreads();
#else
                image_point.x = frame->xmin + col*dx;
                image_point.y = frame->ymin + row*dy;
#endif
                //
                //
                //for(int i = shalos; i < shalos + nhalos; i++)
                //{
                        //IACA_START;
                        //
                        struct point true_coord, true_coord_rot; //, result;
                        //double       R, angular_deviation;
                        complex      zis;
                        //
                        //result.x = result.y = 0.;
                        //
#if 0
                        true_coord.x = img_pt[0] - __ldg(&lens->position_x[i]);
                        true_coord.y = img_pt[1] - __ldg(&lens->position_y[i]);
#else
                        true_coord.x = image_point.x - __ldg(&lens->position_x[i]);
                        true_coord.y = image_point.y - __ldg(&lens->position_y[i]);
#endif
                        double cosi = __ldg(&lens->anglecos[i]);
                        double sinu = __ldg(&lens->anglesin[i]);
                        // positionning at the potential center
                        // Change the origin of the coordinate system to the center of the clump
                        double x = true_coord.x*cosi + true_coord.y*sinu;
                        double y = true_coord.y*cosi - true_coord.x*sinu;
                        //
                        double eps = __ldg(&lens->ellipticity_potential[i]);
                        //
                        double sqe  = sqrt(eps);
                        //
                        double rem2 = x*x/((1. + eps)*(1. + eps)) + y*y/((1. - eps)*(1. - eps));
                        //
                        complex zci;
                        complex znum, zden, zres;
                        double norm;
                        //
                        zci.im  = -0.5*(1. - eps*eps)/sqe;
                        //
                        double rc  = __ldg(&lens->rcore[i]);
                        double cx1  = (1. - eps)/(1. + eps);
                        znum.re = cx1*x;
                        znum.im = 2.*sqe*sqrt(rc*rc + rem2) - y/cx1;
                        //
                        zden.re = x;
                        zden.im = 2.*rc*sqe - y;
                        norm    = (zden.re*zden.re + zden.im*zden.im);     // zis = znum/zden
                        //
                        zis.re  = (znum.re*zden.re + znum.im*zden.im)/norm;
                        zis.im  = (znum.im*zden.re - znum.re*zden.im)/norm;
                        //
                        //
                        double b0  = __ldg(&lens->b0[i]);
                        grad.x += b0*(zres.re*cosi - zres.im*sinu);
                        grad.y += b0*(zres.im*cosi + zres.re*sinu);
                //}
                //IACA_END;
                //
                grid_grad_x[index] += grad.x;
                grid_grad_y[index] += grad.y;
        }
}





/*
   typedef struct point (*halo_func_GPU_t) (const struct point *pImage, const struct Potential_SOA *lens, int shalos, int nhalos);

   __constant__ halo_func_GPU_t halo_func_GPU[100] =
   {
   0, 0, 0, 0, 0, module_potentialDerivatives_totalGradient_5_SOA_GPU, 0, 0, module_potentialDerivatives_totalGradient_8_SOA_GPU,  0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
   0,  module_potentialDerivatives_totalGradient_81_SOA_GPU, 0, 0, 0, 0, 0, 0, 0, 0,
   0, 0, 0, 0, 0, 0, 0, 0, 0, 0
   };
 */

	void
module_potentialDerivatives_totalGradient_SOA_CPU_GPU(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos)
{
	struct point grad, clumpgrad;
	//
	grad.x = clumpgrad.x = 0;
	grad.y = clumpgrad.y = 0;
	int shalos = 0;
	int GRID_SIZE = (nbgridcells + BLOCK_SIZE - 1)/BLOCK_SIZE; // number of blocks
	//
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(GRID_SIZE, GRID_SIZE);	
	int count = nhalos;	
	hipMemset(grid_grad_x, 0, nbgridcells*nbgridcells*sizeof(double));
	hipMemset(grid_grad_y, 0, nbgridcells*nbgridcells*sizeof(double));
	module_potentialDerivatives_totalGradient_8_SOA_GPU<<<dimGrid, dimBlock>>> (grid_grad_x, grid_grad_y, lens_gpu, frame, nbgridcells, shalos, count);
	//grid.x += clumpgrad.x;
	//grad.y += clumpgrad.y;

	//
	//	
	/*
	   while (shalos < nhalos)
	   {

	   int lens_type = lens_cpu->type[shalos];
	   int count     = 1;
	   while (lens_cpu->type[shalos + count] == lens_type) count++;
	//std::cerr << "type = " << lens_type << " " << count << " " << shalos << std::endl;
	//printf ("%d %d %d \n",lens_type,count,shalos);
	//
	clumpgrad = (*halo_func_GPU[lens_type]<<<dimGrid, dimBlock>>> )(lens_gpu, shalos, count);
	//
	grad.x += clumpgrad.x;
	grad.y += clumpgrad.y;
	shalos += count;
	}

	return(grad);
	 */
}


        void
module_potentialDerivatives_totalGradient_SOA_CPU_GPU_v2(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos)
{
        struct point grad, clumpgrad;
        //
        grad.x = clumpgrad.x = 0;
        grad.y = clumpgrad.y = 0;
        int shalos = 0;
        int GRID_SIZE = (nbgridcells + BLOCK_SIZE - 1)/BLOCK_SIZE; // number of blocks
        //
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(GRID_SIZE, GRID_SIZE);
        int count = nhalos;
	//
	hipMemset(grid_grad_x, 0, nbgridcells*nbgridcells*sizeof(double));
	hipMemset(grid_grad_y, 0, nbgridcells*nbgridcells*sizeof(double));
	//
	for (int ii = 0; ii < nhalos; ++ii)
		module_potentialDerivatives_totalGradient_8_SOA_GPU<<<dimGrid, dimBlock>>> (grid_grad_x, grid_grad_y, lens_gpu, frame, nbgridcells, ii, 1);
	
        //grid.x += clumpgrad.x;
        //grad.y += clumpgrad.y;

        //
        //
        /*
           while (shalos < nhalos)
           {

           int lens_type = lens_cpu->type[shalos];
           int count     = 1;
           while (lens_cpu->type[shalos + count] == lens_type) count++;
        //std::cerr << "type = " << lens_type << " " << count << " " << shalos << std::endl;
        //printf ("%d %d %d \n",lens_type,count,shalos);
        //
        clumpgrad = (*halo_func_GPU[lens_type]<<<dimGrid, dimBlock>>> )(lens_gpu, shalos, count);
        //
        grad.x += clumpgrad.x;
        grad.y += clumpgrad.y;
        shalos += count;
        }

        return(grad);
         */
}

