#include "hip/hip_runtime.h"
/**
* @Author Christoph Schaefer, EPFL (christophernstrerne.schaefer@epfl.ch), Gilles Fourestey (gilles.fourestey@epfl.ch)
* @date   July 2017
* @version 0,1
*
*/

#include <fstream>
#include "grid_map_dpl_GPU.cuh"
#include "gradient_GPU.cuh"
//#include "gradient_GPU.cu"
#include <structure_hpc.hpp>

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0


#define cudasafe 

extern "C" 
{
	type_t myseconds();
}



//GPU mapping function declaration to change when figured out linkage problems
__global__ void dpl_grid_GPU(type_t *dpl, type_t dl0s, type_t ds, type_t z,int nbgridcells);
////Map function selection
#if 0
map_gpu_function_t select_map_dpl_function(const struct runmode_param* runmode){

		if(runmode->dpl == 1){
			return &dpl_1_grid_CPU_GPU;
		}
		else{
			fprintf(stderr, "ERROR: Mass mode %d not supported yet \n",runmode->mass);
			exit(-1);
		}

	return 0;
}
#endif

////General Map calculation
void map_grid_dpl_GPU(map_gpu_function_t mapfunction, type_t *grid_grad_x, type_t *grid_grad_y, const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells,int mode_amp, type_t z )
{
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells - 1);
    type_t dy = (frame->ymax - frame->ymin)/(nbgridcells - 1);
    //
    map_grid_dpl_GPU(mapfunction, grid_grad_x, grid_grad_y, cosmo, frame, lens, nhalos,mode_amp,z, dx, dy, nbgridcells, nbgridcells, 0, 0);
}
//
void map_grid_dpl_GPU(map_gpu_function_t mapfunction, type_t *grid_grad_x, type_t *grid_grad_y, const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int mode_amp, type_t z, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	type_t *grid_grad_x_gpu, *grid_grad_y_gpu ;

	type_t dl0s = module_cosmodistances_objectObject(lens->z[0], z, *cosmo);
	type_t dos = module_cosmodistances_observerObject(z, *cosmo);
	type_t dol = module_cosmodistances_observerObject(lens->z[0], *cosmo);

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradientgpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradientgpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"Gradientgpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradientgpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad_x_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradientgpu.cu : Alloc source_x_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad_y_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradientgpu.cu : Alloc source_y_gpu: " );
	//
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradientgpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradientgpu.cu : Copy frame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	//
	cudasafe(hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice), "Gradientgpu.cu: Copy lens_kernel");
	//
	type_t time = -myseconds();
	module_potentialDerivatives_totalGradient_SOA_CPU_GPU(grid_grad_x_gpu, grid_grad_y_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	dpl_grid_CPU_GPU(grid_grad_x_gpu,dl0s,dos,dol,cosmo->h,z,nbgridcells_x,nbgridcells_y,frame);
	dpl_grid_CPU_GPU(grid_grad_y_gpu,dl0s,dos,dol,cosmo->h,z,nbgridcells_x,nbgridcells_y,frame);
	//cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	time += myseconds();
	//std::cout << "	kernel time = " << time << " s." << std::endl;
	//

	cudasafe(hipMemcpy( grid_grad_x, grid_grad_x_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradientgpu.cu : Copy source_x_gpu: " );
	cudasafe(hipMemcpy( grid_grad_y, grid_grad_y_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost)," --- Gradientgpu.cu : Copy source_y_gpu: " );
	//
	//printf("-----> %f %f \n",grid_grad_x[Nx], grid_grad_y[Ny]);
	// Free GPU memory
	hipFree(lens_kernel);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(frame_gpu);
	hipFree(grid_grad_x_gpu);
	hipFree(grid_grad_y_gpu);
}

////Map functions
//DPL NR 1
void dpl_grid_CPU_GPU(type_t *map, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        dpl_grid_GPU<<<grid, threads>>> (map,dl0s,ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "dpl_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void dpl_grid_GPU(type_t *dpl, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	//ellipse amp;
	type_t dlsds= dl0s/ds;
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {

    	int index = row*nbgridcells + col;
        dpl[index] = dpl[index] * dlsds;
        //if(col == 0 and row == 0)printf(" GPUUUU:  Grad  %f %f %f  ABC %f %f %f dlsds %f ab %f %f %f \n",grid_grad2_a[index]*dlsds,grid_grad2_b[index]*dlsds, grid_grad2_c[index]*dlsds,A,B,C,dlsds,amp.a,amp.b,(A - C)*(A - C) + 4*B*B);
    }
}


