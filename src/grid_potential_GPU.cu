#include "hip/hip_runtime.h"
/**
* @Author Christoph Schaefer, EPFL (christophernstrerne.schaefer@epfl.ch), Gilles Fourestey (gilles.fourestey@epfl.ch)
* @date   July 2017
* @version 0,1
*
*/
#include <fstream>
#include "grid_gradient2_GPU.cuh"
#include "gradient2_GPU.cuh"
#include <structure_hpc.hpp>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0


#define cudasafe 

extern "C" 
{
	type_t myseconds();
}

__global__ void
module_potential_totalPotential_SOA_GPU(type_t *potential_GPU, const struct Potential_SOA *lens, const struct grid_param *frame, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart);
////
void
module_potential_SOA_CPU_GPU(type_t *grid_potential, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart);

void
potential_grid_GPU(type_t *grid_potential, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart);
//
//void
//module_potentialDerivatives_totalGradient_SOA_CPU_GPU_v2(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);
//
//
//
void potential_grid_GPU(type_t *grid_potential, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells)
{
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells - 1);
    type_t dy = (frame->ymax - frame->ymin)/(nbgridcells - 1);
        //
    potential_grid_GPU(grid_potential, frame, lens, nhalos, dx, dy, nbgridcells, nbgridcells, 0, 0);
}
//
//
//
void potential_grid_GPU(type_t *grid_potential, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{

	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;
	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	type_t *grid_potential_gpu;

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"grid_potential_GPU.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"grid_potential_GPU.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"grid_potential_GPU.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"grid_potential_GPU.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_potential_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"grid_potential_GPU.cu : Alloc source_a_gpu: " );
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"grid_potential_GPU.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"grid_potential_GPU.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"grid_potential_GPU.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"grid_potential_GPU.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"grid_potential_GPU.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"grid_potential_GPU.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"grid_potential_GPU.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"grid_potential_GPU.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"grid_potential_GPU.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"grid_potential_GPU.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"grid_potential_GPU.cu : Copy frame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	//
	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
	//
	type_t time = -myseconds();
	//
	module_potential_SOA_CPU_GPU(grid_potential_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	//cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	time += myseconds();
	//std::cout << "	kernel time = " << time << " s." << std::endl;
	//

	cudasafe(hipMemcpy( grid_potential, grid_potential_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- grid_potential_GPU.cu : Copy source_a_gpu: " );	//
	//printf("-----> %f %f \n",grid_grad_x[Nx], grid_grad_y[Ny]);
	// Free GPU memory
	hipFree(lens_kernel);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(grid_potential_gpu);
}

void 
module_potential_SOA_CPU_GPU(type_t *grid_potential, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (double) (8*nhalos + BLOCK_SIZE_X*nbgridcells/BLOCK_SIZE_Y)*sizeof(double));
        printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(grid_potential, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        //module_potentialDerivatives_totalGradient_SOA_GPU<<<grid, threads>>> (grid_grad_x, grid_grad_y, lens, frame, nhalos, nbgridcells_x);
        module_potential_totalPotential_SOA_GPU<<<grid, threads>>> (grid_potential,  lens_gpu, frame, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
        cudasafe(hipGetLastError(), "module_potential_totalPotential_SOA_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//


