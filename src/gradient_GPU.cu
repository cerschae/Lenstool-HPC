#include "hip/hip_runtime.h"
/**
* @Author Christoph Schaefer, EPFL (christophernstrerne.schaefer@epfl.ch), Gilles Fourestey (gilles.fourestey@epfl.ch)
* @date   July 2017
* @version 0,1
*
*/
#include <fstream>
#include "grid_gradient_GPU.cuh"
#include "gradient.hpp"
#include "gradient_GPU.cuh"
#include "gradient.hpp"

#define BLOCK_SIZE_X 8
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0

extern "C" {
double myseconds();
}
/*
void
module_potentialDerivatives_totalGradient_SOA_CPU_GPU(type_t *grid_grad_x, type_t *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);

void
module_potentialDerivatives_totalGradient_SOA_CPU_GPU_v2(type_t *grid_grad_x, type_t *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);

void calculate_cossin_values(type_t *theta_cos, type_t *theta_sin, type_t *angles, int nhalos ){
	for(int i = 0 ; i < nhalos; i++)
	{
		theta_cos[i] = cos(angles[i]);
		theta_sin[i] = sin(angles[i]);
	}
}
*/



#if 0

__global__
void
module_potentialDerivatives_totalGradient_8_SOA_GPU_cur(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame, int nbgridcells, int shalos, int nhalos)
{
        //asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
        // 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
        //
        struct point grad, image_point;
	//
        grad.x = 0;
        grad.y = 0;
        //
        int col = blockIdx.x*blockDim.x + threadIdx.x;
        int row = blockIdx.y*blockDim.y + threadIdx.y;
        //
        if ((row < nbgridcells) && (col < nbgridcells))
        {
                //
                int index = row*nbgridcells + col;
                //
                //grid_grad_x[index] = 0.;
                //grid_grad_y[index] = 0.;
                //
                type_t dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
                type_t dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
                //
                image_point.x = frame->xmin + col*dx;
                image_point.y = frame->ymin + row*dy;
		//
                for(int i = shalos; i < shalos + nhalos; i++)
                {
                        //type_t       R, angular_deviation;
                        complex      zis;
                        //
                        // positionning at the potential center
                        // Change the origin of the coordinate system to the center of the clump
                        //
			//@@if ((row == Ny) && (col == Nx)) printf("image_x = %f, %f image_y = %f, %f\n",  image_point.x, frame->xmin, image_point.y,frame->ymin);
			type_t true_coord_x = image_point.x - __ldg(&lens->position_x[i]);
                        type_t true_coord_y = image_point.y - __ldg(&lens->position_y[i]);
			//if ((row == Ny) && (col == Nx)) printf("x = %f y = %f\n",  true_coord_x, true_coord_y);	
			//
                        type_t cosi = __ldg(&lens->anglecos[i]);
                        type_t sinu = __ldg(&lens->anglesin[i]);
			//
                        type_t x = true_coord_x*cosi + true_coord_y*sinu;
                        type_t y = true_coord_y*cosi - true_coord_x*sinu;
			//
			//if ((row == Ny) && (col == Nx)) printf("x = %f y = %f\n",  x, y);	
                        //
                        type_t eps = __ldg(&lens->ellipticity_potential[i]);
                        //
                        type_t sqe  = sqrt(eps);
                        //
                        type_t rem2 = x*x/((1. + eps)*(1. + eps)) + y*y/((1. - eps)*(1. - eps));
                        //
                        complex zci;
                        complex znum, zden, zres;
                        type_t norm;
                        //
			zci.re  = 0;
                        zci.im  = -0.5*(1. - eps*eps)/sqe;
			//@@if ((col == Nx) && (row == Ny)) printf("%d %d, zis: %f %f\n", row, col, zci.re, zci.im);
                        //
                        type_t rc  = __ldg(&lens->rcore[i]);
                        type_t cx1  = (1. - eps)/(1. + eps);
                        znum.re = cx1*x;
                        znum.im = 2.*sqe*sqrt(rc*rc + rem2) - y/cx1;
                        //
                        zden.re = x;
                        zden.im = 2.*rc*sqe - y;
                        norm    = (zden.re*zden.re + zden.im*zden.im);     // zis = znum/zden
			//@@if ((col == Nx) && (row == Ny)) printf("norm = %f\n", norm);
                        //
                        zis.re  = (znum.re*zden.re + znum.im*zden.im)/norm;
                        zis.im  = (znum.im*zden.re - znum.re*zden.im)/norm;
			//
			//@@if ((col == Nx) && (row == Ny)) printf("%d %d, zis: %f %f\n", row, col, zis.re, zis.im);
                        //
                        norm    = zis.re;
                        //
                        zis.re  = log(sqrt(norm*norm + zis.im*zis.im));  // ln(zis) = ln(|zis|)+i.Arg(zis)
                        zis.im  = atan2(zis.im, norm);
			//
			//@@if ((col == Nx) && (row == Ny)) printf("%d %d, zis: %f %f\n", row, col, zis.re, zis.im);
                        //
                        zres.re = zci.re*zis.re - zci.im*zis.im;   // Re( zci*ln(zis) )
                        zres.im = zci.im*zis.re + zis.im*zci.re;   // Im( zci*ln(zis) )
			//
			//@@if ((col == Nx) && (row == Ny)) printf("%d %d, zres: %f %f\n", row, col, zres.re, zres.im);
                        //
                        type_t b0  = __ldg(&lens->b0[i]);
                        grad.x += b0*(zres.re*cosi - zres.im*sinu);
                        grad.y += b0*(zres.im*cosi + zres.re*sinu);
			//@@if ((col == Nx) && (row == Ny)) printf("grad: %f %f\n", grad.x, grad.y);
                }
                //IACA_END;
                //
                grid_grad_x[index] = grad.x;
                grid_grad_y[index] = grad.y;
		//if ((row == 0) && (col == 9)) 
		//printf("%f %f: %f %f\n",  image_point.x, image_point.y, grid_grad_x[index], grid_grad_y[index]);
        }
}

__global__
void
module_potentialDerivatives_totalGradient_8_SOA_GPU_SM2(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame, int nbgridcells, int shalos, int nhalos)
{
	//
        //asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
        // 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
        //
        type_t grad_x, grad_y;
	type_t clumpgrad_x, clumpgrad_y;
	type_t image_point_x, image_point_y;
	//
	SHARED type_t cosi	[200];
	SHARED type_t sinu	[200];
	SHARED type_t rc	[200];
	SHARED type_t b0	[200];
	SHARED type_t epsi	[200];
	SHARED type_t position_x[200];
	SHARED type_t position_y[200];
	SHARED type_t rsqe	[200];
	SHARED type_t sonepeps	[200];
	SHARED type_t sonemeps	[200];
        //
        grad_x = 0;
        grad_y = 0;
        //
        int col = blockIdx.x*blockDim.x + threadIdx.x;
        int row = blockIdx.y*blockDim.y + threadIdx.y;
	int ithread  = threadIdx.y*blockDim.x + threadIdx.x;
	//
	int index = row*nbgridcells + col;
	//
	//grid_grad_x[index] = 0.;
	//grid_grad_y[index] = 0.;
	//
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
	type_t dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
	//
	image_point_x = frame->xmin + col*dx;
	image_point_y = frame->ymin + row*dy;
	//
	int i = ithread;
	if (i < nhalos)
	{
		cosi[i]       = __ldg(&lens->anglecos		  [shalos + i]);
		sinu[i]       = __ldg(&lens->anglesin		  [shalos + i]);
		position_x[i] = __ldg(&lens->position_x		  [shalos + i]);
		position_y[i] = __ldg(&lens->position_y		  [shalos + i]);
		rc[i]         = __ldg(&lens->rcore		  [shalos + i]);
		b0[i]         = __ldg(&lens->b0		          [shalos + i]);
		epsi[i]       = __ldg(&lens->ellipticity_potential[shalos + i]);
		//sonemeps[i]   = 1 - epsi[i];
		//sonepeps[i]   = 1 + epsi[i];
		rsqe[i]	      = sqrt(epsi[i]);
	}
	__syncthreads();
	//
	if ((row < nbgridcells) && (col < nbgridcells))
	{
		for(int i = 0; i < nhalos; i++)
		{
			//
			type_t true_coord_x = image_point_x - position_x[i];
			type_t true_coord_y = image_point_y - position_y[i];
			//
			type_t x = true_coord_x*cosi[i] + true_coord_y*sinu[i];
			type_t y = true_coord_y*cosi[i] - true_coord_x*sinu[i];
			//
			type_t eps     = epsi[i];
			//type_t onemeps = 1 - eps;
			//type_t onepeps = 1 + eps;
			//
			//type_t eps     = epsi[i];
			type_t onemeps = sonemeps[i];
			type_t onepeps = sonepeps[i];
			//
			//type_t sqe  = sqrt(eps);
			type_t sqe  = rsqe[i];
			type_t rem2 = x*x/(onepeps*onepeps) + y*y/(onemeps*onemeps);
			//
			complex      zis;
			//
			type_t znum_re, znum_im;
			type_t zres_re, zres_im;
			type_t norm;
			type_t zden_re, zden_im;
			type_t  zis_re,  zis_im;
			//
			type_t zci_im  = -0.5*(1. - eps*eps)/sqe;
			//
			type_t cx1  = onemeps/onepeps;
			//
			znum_re = cx1*x;
			znum_im = 2.*sqe*sqrt(rc[i]*rc[i] + rem2) - y/cx1;
			//
			zden_re = x;
			zden_im = 2.*rc[i]*sqe - y;
			//
			norm    = (x*x + zden_im*zden_im);     // zis = znum/zden
			zis.re  = (znum_re*x + znum_im*zden_im)/norm;
			zis.im  = (znum_im*x - znum_re*zden_im)/norm;
			//
			norm    = zis.re;
			//
			zis.re  = log(sqrt(norm*norm + zis.im*zis.im));  // ln(zis) = ln(|zis|)+i.Arg(zis)
			zis.im  = atan2(zis.im, norm);
			//
			zres_re = - zci_im*zis.im;   // Re( zci*ln(zis) )
			zres_im =   zci_im*zis.re;   // Im( zci*ln(zis) )
			//
			grad_x += b0[i]*(zres_re*cosi[i] - zres_im*sinu[i]);
			grad_y += b0[i]*(zres_im*cosi[i] + zres_re*sinu[i]);
		}
		//
		grid_grad_x[index] = grad_x;
		grid_grad_y[index] = grad_y;
		//__syncthreads();
	}
}
//
//
//
__global__
void
module_potentialDerivatives_totalGradient_8_SOA_GPU_SM3(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA
 *lens, const struct grid_param *frame, int nbgridcells, int shalos, int nhalos)
{
        //
        //asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
        // 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
        //
        type_t grad_x, grad_y;
        type_t clumpgrad_x, clumpgrad_y;
        type_t image_point_x, image_point_y;
        //
        SHARED type_t cosi      [200];
        SHARED type_t sinu      [200];
        SHARED type_t rci       [200];
        SHARED type_t b0        [200];
        SHARED type_t epsi      [200];
        SHARED type_t position_x[200];
        SHARED type_t position_y[200];
        SHARED type_t rsqe      [200];
	//SHARED type_t sgrad_x   [(BLOCK_SIZE_X + 1)*BLOCK_SIZE_Y];
	//SHARED type_t sgrad_y   [(BLOCK_SIZE_X + 1)*BLOCK_SIZE_Y];

        //SHARED type_t sonepeps  [200];
        //SHARED type_t sonemeps  [200];
        //
        grad_x         = 0;
        grad_y 	       = 0;
        //
        int row        = blockIdx.y*blockDim.y + threadIdx.y;
        int col        = blockIdx.x*blockDim.x + threadIdx.x;
	//
	//int loc_row    = threadIdx.x;
	//int loc_col    = threadIdx.y*blockDim.x + threadIdx.x;
	//
        //int grid_size  = nbgridcells/blockDim.y; 
	//
	//if (threadIdx.x == 0) printf("%d %d %d: row = %d, col = %d, grid_size = %d\n", blockIdx.y, gridDim.y, threadIdx.y, row, col, grid_size);
        //
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
	type_t dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
	//if (threadIdx.x == 0) printf("dx = %f, dy = %f\n", dx, dy);
	//
	image_point_x = frame->xmin + col*dx;
	image_point_y = frame->ymin + row*dy;
	//
	//int iloc  = threadIdx.x*blockDim.y + threadIdx.y;
	int iglob = row*nbgridcells + col;
	int numThreads = blockDim.x*blockDim.y;
	//
	for (int i = 0; i < (nhalos + numThreads - 1)/numThreads; ++i)
	{ 
		int iloc  = threadIdx.y*blockDim.x + threadIdx.x + i*numThreads;
		if (iloc < nhalos)
		{
			cosi[iloc]       = __ldg(&lens->anglecos             [shalos + iloc]);
			sinu[iloc]       = __ldg(&lens->anglesin             [shalos + iloc]);
			position_x[iloc] = __ldg(&lens->position_x           [shalos + iloc]);
			position_y[iloc] = __ldg(&lens->position_y           [shalos + iloc]);
			rci[iloc]        = __ldg(&lens->rcore                [shalos + iloc]);
			b0[iloc]         = __ldg(&lens->b0                   [shalos + iloc]);
			epsi[iloc]       = __ldg(&lens->ellipticity_potential[shalos + iloc]);
			rsqe[iloc]       = sqrt(epsi[iloc]);
		}
	}
	__syncthreads();
	//
	if ((row < nbgridcells) && (col < nbgridcells))
	{
		//
		for(int i = 0; i < nhalos; i++)
		{
			//int index  = iloc; 
#if 1
			type_t rc      = rci[i];
			type_t eps     = epsi[i];
			type_t onemeps = 1 - eps;
			type_t onepeps = 1 + eps;
			//
			type_t sqe = rsqe[i];
			type_t cx1 = onemeps/onepeps;
			//
			//
			//type_t zci_im = 1;
			type_t zci_im  = -0.5*(1. - eps*eps)/sqe;
			type_t inv_onepeps = 1./(onepeps*onepeps);
			type_t inv_onemeps = 1./(onemeps*onemeps);
#endif
			//
			{
				//KERNEL_8;
				type_t grad_x = grad_y = 0.;
				type_t true_coord_y = image_point_y - position_y[i];
				type_t true_coord_x = image_point_x - position_x[i];
				KERNEL_8_reg(0);
				grid_grad_x[iglob +  0] += grad_x;
				grid_grad_y[iglob +  0] += grad_y;
			}
			/*
			{
				//KERNEL_8;
				type_t grad_x = grad_y = 0.;
				type_t true_coord_y = image_point_y - position_y[i];
				type_t true_coord_x = image_point_x - position_x[i] + BLOCK_SIZE_X/2*dx;
				KERNEL_8_reg(0);
				grid_grad_x[iglob + BLOCK_SIZE_X/2] += grad_x;
				grid_grad_y[iglob + BLOCK_SIZE_X/2] += grad_y;
			}
			*/
		//
		}
	}
}
//
//
//
__global__
void
module_potentialDerivatives_totalGradient_8_SOA_GPU_SM4(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA
		lens, const struct grid_param *frame, int nbgridcells, int shalos, int nhalos/*, type_t* dtimer*/)
{
	//
	//asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
	// 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
	//
	type_t grad_x, grad_y;
	type_t clumpgrad_x, clumpgrad_y;
	type_t image_point_x, image_point_y;
	//
	//
	type_t* cosi       = &shared[0*nhalos];
	type_t* sinu       = &shared[1*nhalos];
	type_t* rc         = &shared[2*nhalos];
	type_t* b0         = &shared[3*nhalos];
	type_t* epsi       = &shared[4*nhalos];
	type_t* position_x = &shared[5*nhalos];
	type_t* position_y = &shared[6*nhalos];
	type_t* rsqe       = &shared[7*nhalos];

	//SHARED type_t sonepeps  [200];
	//SHARED type_t sonemeps  [200];
	//
	grad_x         = 0;
	grad_y         = 0;
	//
	int grid_size  =  nbgridcells/gridDim.y;
	int row        =  blockIdx.x*blockDim.x + threadIdx.x;
	int col        = (blockIdx.y*blockDim.y + threadIdx.y)/**grid_size*/;
	//
	//

#if 0
	//SHARED double sgrad_x [
	if (/*(threadIdx.x == 0) &&*/ (blockIdx.x == 0))
	{
		if (threadIdx.x == 0) printf("blockDim.x = %d, blockIdx.x = %d grimdDim.x = %d threadIdx.x = %d\n", blockDim.x, blockIdx.x, gridDim.x, threadIdx.x);
		if (threadIdx.x == 0) printf("blockDim.y = %d, blockIdx.y = %d grimdDim.y = %d threadIdx.y = %d\n", blockDim.y, blockIdx.y, gridDim.y, threadIdx.y);
		if (threadIdx.x == 0) printf("row = %d, col = %d, grid_size = %d\n", row, col, grid_size);
	}
	__syncthreads();
#endif
	//type_t* sgrad_x    = &shared[8*nhalos];
	type_t* sgrad_y    = &shared[8*nhalos + (grid_size + 1)*blockDim.x];
	//
	//
	//grid_grad_x[index] = 0.;
	//grid_grad_y[index] = 0.;
	//
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
	type_t dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
	//
	//if (threadIdx.x == 0) printf("dx = %f, dy = %f\n", dx, dy);
	//
	image_point_x = frame->xmin + col*dx;
	image_point_y = frame->ymin + row*dy;
	return;
	//
	//int i = 0;
#if 0
	for (; i < nhalos; i = i + blockDim.x)	
	{
		int pos = threadIdx.x + i;
		/*if ((threadIdx.x == 0) && (blockIdx.x == 0))*/ printf("pos = %d\n"); 
		__syncthreads();
		//
		cosi[pos]       = __ldg(&lens->anglecos             [shalos + pos]);
		sinu[pos]       = __ldg(&lens->anglesin             [shalos + pos]);
		position_x[pos] = __ldg(&lens->position_x           [shalos + pos]);
		position_y[pos] = __ldg(&lens->position_y           [shalos + pos]);
		rc[pos]         = __ldg(&lens->rcore                [shalos + pos]);
		b0[pos]         = __ldg(&lens->b0                   [shalos + pos]);
		epsi[pos]       = __ldg(&lens->ellipticity_potential[shalos + pos]);
		rsqe[pos]       = sqrt(epsi[i]);
		//
	}
#endif
#if 0
	if (threadIdx.x == 0)
		for (; i < nhalos; i += 1)
		{
			cosi[i]       = __ldg(&lens->anglecos             [shalos + i]);
			sinu[i]       = __ldg(&lens->anglesin             [shalos + i]);
			position_x[i] = __ldg(&lens->position_x           [shalos + i]);
			position_y[i] = __ldg(&lens->position_y           [shalos + i]);
			rc[i]         = __ldg(&lens->rcore                [shalos + i]);
			b0[i]         = __ldg(&lens->b0                   [shalos + i]);
			epsi[i]       = __ldg(&lens->ellipticity_potential[shalos + i]);
			rsqe[i]       = sqrt(epsi[i]);
		}
#endif
	__syncthreads();
	//if ((row == col == 0)) printf("shared mem done...\n");
	//
	if (row < nbgridcells)
	{
		//for(int icol = 0; icol < grid_size; ++icol){
                //      if (col + icol < nbgridcells){
                //grad_x = grad_y = 0.;
		int  index  = row*nbgridcells + col;
                //
                for(int i = 0; i < nhalos; i++)
                {
                        int sindex  = threadIdx.x*grid_size; 
#if 0
                        type_t eps     = epsi[i];
                        type_t onemeps = 1 - eps;
                        type_t onepeps = 1 + eps;
                        //
                        type_t sqe = rsqe[i];
                        type_t cx1 = onemeps/onepeps;
                        //
                        //
                        //type_t x = true_coord_y*sinu[i];
                        //type_t y = true_coord_y*cosi[i];
                        type_t true_coord_y = image_point_y - position_y[i];
                        type_t true_coord_x = image_point_x - position_x[i] /*+ icol*dx*/;
                        //
                        complex zci;
                        zci.im  = -0.5*(1. - eps*eps)/sqe;
                        type_t inv_onepeps = 1./(onepeps*onepeps);
                        type_t inv_onemeps = 1./(onemeps*onemeps);
#endif
                        //
                        for(int icol = 0; icol < grid_size; ++icol)
                        {
                                if (col + icol < nbgridcells)
                                {
#if 0
                                        if ((row == 1) && (col == 1)) printf("%d %d: %f %f\n", row, col, true_coord_x, true_coord_y);

                                        true_coord_x = image_point_x - position_x[i] + icol*dx;
                                        //
                                        //x += true_coord_x*cosi[i];
                                        //y -= true_coord_x*sinu[i];
                                        type_t x = true_coord_x*cosi[i] + true_coord_y*sinu[i];
                                        type_t y = true_coord_y*cosi[i] - true_coord_x*sinu[i];
                                        //
                                        //if ((row == 1) && (col == 0)) printf("i = %d, eps = %f\n", i, eps);
                                        //
                                        //double eps     = epsi[i];
                                        //double onemeps = sonemeps[i];
                                        //double onepeps = sonepeps[i];
                                        //
                                        //double sqe  = sqrt(eps);
                                        //double rem2 = x*x/(onepeps*onepeps) + y*y/(onemeps*onemeps);
                                        type_t rem2 = x*x*inv_onepeps + y*y*inv_onemeps;
                                        //
                                        //
                                        //double znum_re, znum_im;
                                        //double zres_re, zres_im;
                                        //double zden_re, zden_im;
                                        //double  zis_re,  zis_im;
                                        type_t norm;
                                        //
                                        complex      zis;
                                        complex      znum;
                                        complex      zden;
                                        complex      zres;
                                        //
                                        //double cx1  = onemeps/onepeps;
                                        //
                                        znum.re = cx1*x;
                                        znum.im = 2.*sqe*sqrt(rc[i]*rc[i] + rem2) - y/cx1;
                                        //
                                        zden.re = x;
                                        zden.im = 2.*rc[i]*sqe - y;
                                        //
                                        norm    = (x*x + zden.im*zden.im);     // zis = znum/zden
                                        zis.re  = (znum.re*x + znum.im*zden.im)/norm;
                                        zis.im  = (znum.im*x - znum.re*zden.im)/norm;
                                        //
                                        norm    = zis.re;
                                        //
                                        zis.re  = log(sqrt(norm*norm + zis.im*zis.im));  // ln(zis) = ln(|zis|)+i.Arg(zis)
                                        zis.im  = atan2(zis.im, norm);
                                        //
                                        zres.re = - zci.im*zis.im;   // Re( zci*ln(zis) )
                                        zres.im =   zci.im*zis.re;   // Im( zci*ln(zis) )
                                        //
                                        grid_grad_x[index] += b0[i]*(zres.re*cosi[i] - zres.im*sinu[i]);
                                        grid_grad_y[index] += b0[i]*(zres.im*cosi[i] + zres.re*sinu[i]);
#endif
                                        //sgrad_x[sindex] += (float)  sindex;
                                        sgrad_y[sindex] += (float) -sindex;
                                        //sindex++;
                                        //
                                        //grid_grad_x[index] += grad_x;
                                        //grid_grad_y[index] += grad_y;
                                }
                                //
                        }
                }
                        __syncthreads();
	return;
                //
#if 0
		int sindex = threadIdx.x*grid_size;
		for(int icol = 0; icol < grid_size; ++icol)
		{
			if (col + icol < nbgridcells)
			{
                		grid_grad_x[index + col] = sgrad_x[sindex];
                		grid_grad_y[index + col] = sgrad_y[sindex];
				sindex++;
			}
		}
		__syncthreads();
#endif
        }
}



__global__
void
module_potentialDerivatives_totalGradient_8_SOA_GPU_v2(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame, int nbgridcells, int i, int nhalos)
{
	//asm volatile("# module_potentialDerivatives_totalGradient_SOA begins");
	// 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
	//
	struct point grad, image_point;
	grad.x = 0;
	grad.y = 0;
	//
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//
	if ((row < nbgridcells) && (col < nbgridcells))
	{
		//
		int index = col*nbgridcells + row;
		//
		//grid_grad_x[index] = 0.;
		//grid_grad_y[index] = 0.;
		//
		type_t dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
		type_t dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
		//
#if 0
		/*SHARED*/ type_t img_pt[2];
		if ((row == 0) && (col == 0))
		{
			img_pt[0] = frame->xmin + col*dx;
			img_pt[1] = frame->ymin + row*dy;
		}
		__syncthreads();
#else
		image_point.x = frame->xmin + col*dx;
		image_point.y = frame->ymin + row*dy;
#endif
		//
		//
		//for(int i = shalos; i < shalos + nhalos; i++)
		//{
		//IACA_START;
		//
		struct point true_coord; //, result;
		//type_t       R, angular_deviation;
		complex      zis;
		//
		//result.x = result.y = 0.;
		//
#if 0
		true_coord.x = img_pt[0] - __ldg(&lens->position_x[i]);
		true_coord.y = img_pt[1] - __ldg(&lens->position_y[i]);
#else
		true_coord.x = image_point.x - __ldg(&lens->position_x[i]);
		true_coord.y = image_point.y - __ldg(&lens->position_y[i]);
#endif
		type_t cosi = __ldg(&lens->anglecos[i]);
		type_t sinu = __ldg(&lens->anglesin[i]);
		// positionning at the potential center
		// Change the origin of the coordinate system to the center of the clump
		type_t x = true_coord.x*cosi + true_coord.y*sinu;
		type_t y = true_coord.y*cosi - true_coord.x*sinu;
		//
		type_t eps = __ldg(&lens->ellipticity_potential[i]);
		//
		type_t sqe  = sqrt(eps);
		//
		type_t rem2 = x*x/((1. + eps)*(1. + eps)) + y*y/((1. - eps)*(1. - eps));
		//
		complex zci;
		complex znum, zden, zres;
		type_t norm;
		//
		zci.im  = -0.5*(1. - eps*eps)/sqe;
		//
		type_t rc  = __ldg(&lens->rcore[i]);
		type_t cx1  = (1. - eps)/(1. + eps);
		znum.re = cx1*x;
		znum.im = 2.*sqe*sqrt(rc*rc + rem2) - y/cx1;
		//
		zden.re = x;
		zden.im = 2.*rc*sqe - y;
		norm    = (zden.re*zden.re + zden.im*zden.im);     // zis = znum/zden
		//
		zis.re  = (znum.re*zden.re + znum.im*zden.im)/norm;
		zis.im  = (znum.im*zden.re - znum.re*zden.im)/norm;
		//
		type_t b0  = __ldg(&lens->b0[i]);
		//grad.x += b0*(zres.re*cosi - zres.im*sinu);
		//grad.y += b0*(zres.im*cosi + zres.re*sinu);
		//
		grid_grad_x[index] += grad.x;
		grid_grad_y[index] += grad.y;
	}
}

#endif

__device__ point module_potentialDerivatives_totalGradient_5_SOA_GPU(const struct point *pImage, const struct Potential_SOA *lens, int shalos, int nhalos){
    //asm volatile("# module_potentialDerivatives_totalGradient_SIS_SOA_v2 begins");
    //printf("# module_potentialDerivatives_totalGradient_SIS_SOA_v2 begins\n");
    //
    struct point grad, result;
    grad.x = 0;
    grad.y = 0;
    for(int i = shalos; i < shalos + nhalos; i++)
    {
            //
		struct point true_coord;
		//
		true_coord.x = pImage->x - lens->position_x[i];
		true_coord.y = pImage->y - lens->position_y[i];
		//
		//true_coord_rotation = rotateCoordinateSystem(true_coord, lens->ellipticity_angle[i]);
		type_t cose = lens->anglecos[i];
		type_t sine = lens->anglesin[i];
		//
		type_t x = true_coord.x*cose + true_coord.y*sine;
		type_t y = true_coord.y*cose - true_coord.x*sine;
		//
		type_t ell_pot = lens->ellipticity_potential[i];
		//
		type_t b0_inv_R = lens->b0[i]/sqrt(x*x*(1 - ell_pot) + y*y*(1 + ell_pot));
		//
		result.x = (1 - ell_pot)*x*b0_inv_R;
		result.y = (1 + ell_pot)*y*b0_inv_R;
		//
		grad.x += result.x*cose - result.y*sine;
		grad.y += result.y*cose + result.x*sine;
    }
    return grad;

}
__device__ point module_potentialDerivatives_totalGradient_8_SOA_GPU(const struct point *image_point, const struct Potential_SOA *lens, int shalos, int nhalos){
	struct point grad;
	grad.x = 0;
	grad.y = 0;
	//
	for(int i = shalos; i < shalos + nhalos; i++)
	{
        complex      zis;
        // positioning at the potential center
        // Change the origin of the coordinate system to the center of the clump
        //@@if ((row == Ny) && (col == Nx)) printf("image_x = %f, %f image_y = %f, %f\n",  image_point.x, frame->xmin, image_point.y,frame->ymin);

        type_t true_coord_x = image_point->x - __ldg(&lens->position_x[i]);
        type_t true_coord_y = image_point->y - __ldg(&lens->position_y[i]);
        //if ((row == Ny) && (col == Nx)) printf("x = %f y = %f\n",  true_coord_x, true_coord_y);
        //
        type_t cosi = __ldg(&lens->anglecos[i]);
        type_t sinu = __ldg(&lens->anglesin[i]);
        //
        type_t x = true_coord_x*cosi + true_coord_y*sinu;
        type_t y = true_coord_y*cosi - true_coord_x*sinu;
        //
        //if ((row == Ny) && (col == Nx)) printf("x = %f y = %f\n",  x, y);
        //
        type_t eps = __ldg(&lens->ellipticity_potential[i]);
        //
        type_t sqe  = sqrt(eps);
        //
        type_t rem2 = x*x/((1. + eps)*(1. + eps)) + y*y/((1. - eps)*(1. - eps));
        //
        complex zci;
        complex znum, zden, zres;
        type_t norm;
        //
        zci.re  = 0;
        zci.im  = -0.5*(1. - eps*eps)/sqe;
        //@@if ((col == Nx) && (row == Ny)) printf("%d %d, zis: %f %f\n", row, col, zci.re, zci.im);
        //
        type_t rc  = __ldg(&lens->rcore[i]);
        type_t cx1  = (1. - eps)/(1. + eps);

        znum.re = cx1*x;
        znum.im = 2.*sqe*sqrt(rc*rc + rem2) - y/cx1;
        //
        zden.re = x;
        zden.im = 2.*rc*sqe - y;
        norm    = (zden.re*zden.re + zden.im*zden.im);     // zis = znum/zden
        //@@if ((col == Nx) && (row == Ny)) printf("norm = %f\n", norm);
        //
        zis.re  = (znum.re*zden.re + znum.im*zden.im)/norm;
        zis.im  = (znum.im*zden.re - znum.re*zden.im)/norm;
        //
        //@@if ((col == Nx) && (row == Ny)) printf("%d %d, zis: %f %f\n", row, col, zis.re, zis.im);
        //
        norm    = zis.re;
        //
        zis.re  = log(sqrt(norm*norm + zis.im*zis.im));  // ln(zis) = ln(|zis|)+i.Arg(zis)
        zis.im  = atan2(zis.im, norm);
        //
        //@@if ((col == Nx) && (row == Ny)) printf("%d %d, zis: %f %f\n", row, col, zis.re, zis.im);
        //
        zres.re = zci.re*zis.re - zci.im*zis.im;   // Re( zci*ln(zis) )
        zres.im = zci.im*zis.re + zis.im*zci.re;   // Im( zci*ln(zis) )
        //
        //@@if ((col == Nx) && (row == Ny)) printf("%d %d, zres: %f %f\n", row, col, zres.re, zres.im);
        //
        type_t b0  = __ldg(&lens->b0[i]);

        grad.x += b0*(zres.re*cosi - zres.im*sinu);
        grad.y += b0*(zres.im*cosi + zres.re*sinu);
        //@@if ((col == Nx) && (row == Ny)) printf("grad: %f %f\n", grad.x, grad.y);
	}
	//IACA_END;
	//
	return(grad);

}
__device__ point module_potentialDerivatives_totalGradient_81_SOA_GPU(const struct point *pImage, const struct Potential_SOA *lens, int shalos, int nhalos){
    //asm volatile("# module_potentialDerivatives_totalGradient_81_SOA begins");
    //std::cout << "# module_potentialDerivatives_totalGradient_81_SOA begins" << std::endl;
    // 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
    //
    struct point grad;
    grad.x = 0;
    grad.y = 0;
    for(int i = shalos; i < shalos + nhalos; i++)
    {
		//IACA_START;
		//
		struct point true_coord; //, result;
		//type_t       R, angular_deviation;
		complex      zis;
		//
		//result.x = result.y = 0.;
		//
		true_coord.x = pImage->x - lens->position_x[i];
		true_coord.y = pImage->y - lens->position_y[i];
		/*positionning at the potential center*/
		// Change the origin of the coordinate system to the center of the clump
		type_t cose = lens->anglecos[i];
		type_t sine = lens->anglesin[i];
		type_t x = true_coord.x*cose + true_coord.y*sine;
		type_t y = true_coord.y*cose - true_coord.x*sine;
		//
		type_t eps  = lens->ellipticity_potential[i];
		type_t rc   = lens->rcore[i];
		type_t rcut = lens->rcut[i];
		type_t b0   = lens->b0[i];
		type_t t05  = b0*rcut/(rcut - rc);
		//
		type_t sqe  = sqrt(eps);
		//
		type_t cx1  = (1. - eps)/(1. + eps);
		type_t cxro = (1. + eps)*(1. + eps);
		type_t cyro = (1. - eps)*(1. - eps);
		//
		type_t rem2 = x*x/cxro + y*y/cyro;
		//
		complex zci, znum, zden, zres_rc, zres_rcut;
		type_t norm;
		//
		zci.re  = 0;
		zci.im  = -0.5*(1. - eps*eps)/sqe;
		//
		// step 1
		{
		KERNEL(rc, zres_rc)
		}
		// step 2
		{
		KERNEL(rcut, zres_rcut)
		}
		zis.re  = t05*(zres_rc.re - zres_rcut.re);
		zis.im  = t05*(zres_rc.im - zres_rcut.im);
		// rotation
		grad.x += (zis.re*cose - zis.im*sine);
		grad.y += (zis.im*cose + zis.re*sine);
            //
    }
    //
    return(grad);

}

#if 1
typedef struct point (*halo_func_GPU_t) (const struct point *pImage, const struct Potential_SOA *lens, int shalos, int nhalos);

__constant__ halo_func_GPU_t halo_func_GPU[100] =
{
	0, 0, 0, 0, 0, module_potentialDerivatives_totalGradient_5_SOA_GPU, 0, 0, module_potentialDerivatives_totalGradient_8_SOA_GPU,  0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	   0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	   0,  module_potentialDerivatives_totalGradient_81_SOA_GPU, 0, 0, 0, 0, 0, 0, 0, 0,
	   0, 0, 0, 0, 0, 0, 0, 0, 0, 0
	   };
#endif

__global__
void
module_potentialDerivatives_totalGradient_SOA_GPU(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
        struct point grad, clumpgrad, image_point;
        //
        int col = blockIdx.x*blockDim.x + threadIdx.x;
        int row = blockIdx.y*blockDim.y + threadIdx.y;

        //
        if ((row + 0*jstart < nbgridcells_y) && (col + 0*istart < nbgridcells_x))
        {
                //
                //double dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
                //double dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
                //
                int index = row*nbgridcells_x + col;
                // Create temp grad variable to minimise writing to global memory grid_grad
                grad.x = 0.;
                grad.y = 0.;
                //
                image_point.x = frame->xmin + (col + istart)*dx;
                image_point.y = frame->ymin + (row + jstart)*dy;
                //
                int shalos = 0;
                while (shalos < nhalos)
                {
                        int lens_type = lens->type[shalos];
                        int count     = 1;
                        while (lens->type[shalos + count] == lens_type) count++;
                        //
                        //if(row == 0 && col == 0) printf("type = %d, count %d , shalos %d \n", lens_type,count,shalos );
                        //
                        clumpgrad = (*halo_func_GPU[lens_type])(&image_point, lens, shalos, count);
                        //
                        grad.x += clumpgrad.x;
                        grad.y += clumpgrad.y;
                        shalos += count;
                }
                // Write to global memory
                grid_grad_x[index] = grad.x;
                grid_grad_y[index] = grad.y;
                //if ((row == 0) && (col == 9))
                //printf("%f %f: %f %f\n",  image_point.x, image_point.y, grid_grad_x[index], grid_grad_y[index]);
        }
}

__global__
void
module_potentialDerivatives_totalGradient_SOA_GPU(type_t *grid_grad_x, type_t *grid_grad_y, const struct Potential_SOA *lens, const struct grid_param *frame, int nbgridcells, int nhalos)
{
    struct point grad, clumpgrad, image_point;
    //
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
        //
        type_t dx = (frame->xmax - frame->xmin)/(nbgridcells-1);
        type_t dy = (frame->ymax - frame->ymin)/(nbgridcells-1);
		//
		int index = row*nbgridcells + col;
		// Create temp grad variable to minimise writing to global memory grid_grad
		grad.x = 0;
		grad.y = 0;
		//
		image_point.x = frame->xmin + col*dx;
		image_point.y = frame->ymin + row*dy;
		//
		int shalos = 0;
		while (shalos < nhalos)
		{
			int lens_type = lens->type[shalos];
			int count     = 1;
			while (lens->type[shalos + count] == lens_type) count++;
			//
			//if(row == 0 && col == 0) printf("type = %d, count %d , shalos %d \n", lens_type,count,shalos );
			//
			clumpgrad = (*halo_func_GPU[lens_type])(&image_point, lens, shalos, count);
			//
			grad.x += clumpgrad.x;
			grad.y += clumpgrad.y;
			shalos += count;
		}
		// Write to global memory
		grid_grad_x[index] = grad.x;
		grid_grad_y[index] = grad.y;

	//if ((row == 0) && (col == 9))
	//printf("%f %f: %f %f\n",  image_point.x, image_point.y, grid_grad_x[index], grid_grad_y[index]);
    }

}
