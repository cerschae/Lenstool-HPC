#include "hip/hip_runtime.h"
/**
Lenstool-HPC: HPC based massmodeling software and Lens-map generation
Copyright (C) 2017  Christoph Schaefer, EPFL (christophernstrerne.schaefer@epfl.ch), Gilles Fourestey (gilles.fourestey@epfl.ch)

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

@brief: Function for second order derivative computation over a grid

*/


#include <fstream>
#include "grid_gradient2_GPU.cuh"
#include "gradient2_GPU.cuh"
#include <structure_hpc.hpp>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0


#define cudasafe 

extern "C" 
{
	type_t myseconds();
}

__global__ void module_potentialDerivatives_totalGradient2_SOA_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d,  const struct Potential_SOA *lens, const struct grid_param *frame, int nbgridcells, int nhalos);

////
void
module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(type_t *grid_grad2_a, type_t *grid_grad_b, type_t *grid_grad2_c, type_t *grid_grad_d, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);
//
void gradient2_grid_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart);
//
//void
//module_potentialDerivatives_totalGradient_SOA_CPU_GPU_v2(double *grid_grad_x, double *grid_grad_y, const struct grid_param *frame, const struct Potential_SOA *lens_cpu, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos);
//
//
//
void gradient2_grid_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells)
{
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells - 1);
    type_t dy = (frame->ymax - frame->ymin)/(nbgridcells - 1);
        //
    gradient2_grid_GPU(grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d, frame, lens, nhalos, dx, dy, nbgridcells, nbgridcells, 0, 0);
}
//
//
//
void gradient2_grid_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{

	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;

	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

/*
	if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
	{
		fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
		exit(-1);
	}
	else
	{
		nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
		// per Block that the GPU supports
	}
*/
	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	type_t *grid_grad2_a_gpu, *grid_grad2_b_gpu , *grid_grad2_c_gpu, *grid_grad2_d_gpu;

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradient2gpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradient2gpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradient2gpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_a_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_a_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_b_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_b_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_c_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_c_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_d_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_d_gpu: " );
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2pu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy fame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	//
	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
	//
	type_t time = -myseconds();
	//module_potentialDerivatives_totalGradient_SOA_CPU_GPU(grid_grad_x_gpu, grid_grad_y_gpu, frame_gpu, lens_kernel, nbgridcells_x, nhalos);
	module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	//cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	time += myseconds();
	//std::cout << "	kernel time = " << time << " s." << std::endl;
	//

	cudasafe(hipMemcpy( grid_grad2_a, grid_grad2_a_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_a_gpu: " );
	cudasafe(hipMemcpy( grid_grad2_b, grid_grad2_b_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost)," --- Gradient2gpu.cu : Copy source_b_gpu: " );
	cudasafe(hipMemcpy( grid_grad2_c, grid_grad2_c_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_c_gpu: " );
	cudasafe(hipMemcpy( grid_grad2_d, grid_grad2_d_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost)," --- Gradient2gpu.cu : Copy source_d_gpu: " );
	//
	//printf("-----> %f %f \n",grid_grad_x[Nx], grid_grad_y[Ny]);
	// Free GPU memory
	hipFree(lens_kernel);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(grid_grad2_a_gpu);
	hipFree(grid_grad2_b_gpu);
	hipFree(grid_grad2_c_gpu);
	hipFree(grid_grad2_d_gpu);
}

void 
module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (double) (8*nhalos + BLOCK_SIZE_X*nbgridcells/BLOCK_SIZE_Y)*sizeof(double));
        printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(grid_grad2_a, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_b, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_c, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_d, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        //module_potentialDerivatives_totalGradient_SOA_GPU<<<grid, threads>>> (grid_grad_x, grid_grad_y, lens, frame, nhalos, nbgridcells_x);
        module_potentialDerivatives_totalGradient2_SOA_GPU<<<grid, threads>>> (grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,  lens_gpu, frame, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
        cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU_8_SOA_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
//
void
module_potentialDerivatives_Kmap_SOA_CPU_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (double) (8*nhalos + BLOCK_SIZE_X*nbgridcells/BLOCK_SIZE_Y)*sizeof(double));
        printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(grid_grad2_a, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_b, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_c, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_d, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        //module_potentialDerivatives_totalGradient_SOA_GPU<<<grid, threads>>> (grid_grad_x, grid_grad_y, lens, frame, nhalos, nbgridcells_x);
        module_potentialDerivatives_Kmap_SOA_GPU<<<grid, threads>>> (grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,  lens_gpu, frame, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
        cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU_8_SOA_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//


