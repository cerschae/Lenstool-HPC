#include "hip/hip_runtime.h"
/**
 * @Author Christoph Schaefer, EPFL (christophernstrerne.schaefer@epfl.ch), Gilles Fourestey (gilles.fourestey@epfl.ch)
 * @date   July 2017
 * @version 0,1
 *
 */
#include <fstream>
#include "potential_GPU.cuh"
#include "structure_hpc.hpp"

#define BLOCK_SIZE_X 8
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0

extern "C" {
double myseconds();
}

__device__ double  pi05_gpu(double x, double y, double eps, double rc, double b0)
{
    double  sqe, ci, cxro, cyro, rem2, e1, e2, z;
    complex eta, zeta, b1, b2, a1, a2, c1, c2, ckk;


    sqe = sqrt(eps);
    ci = .5 * (1. - eps * eps) / sqe;
    cxro = (1. + eps) * (1. + eps);
    cyro = (1. - eps) * (1. - eps);
    rem2 = x * x / cxro + y * y / cyro;
    e1 = 2.*sqe / (1 - eps);
    e2 = 2.*sqe / (1 + eps);
    z = sqrt(x * x + y * y);
    eta = cpx_GPU(-.5 * asinh(e1 * y / z), .5 * asin(e2 * x / z));
    zeta = cpx_GPU( 0.5 * log( (sqrt(rem2) + sqrt(rc * rc + rem2)) / rc), 0. );
    b1 = coshcpx_GPU(acpx_GPU(eta, zeta));
    b2 = coshcpx_GPU(scpx_GPU(eta, zeta));
    a1 = lncpx_GPU( dcpx_GPU(sqcpx_GPU(coshcpx_GPU(eta)), pcpx_GPU(b1, b2)) );
    a2 = lncpx_GPU(dcpx_GPU(b1, b2));
    c1 = pcpx_GPU(sinhcpx_GPU(pcpxflt_GPU(eta, 2.)), a1);
    c2 = pcpx_GPU(sinhcpx_GPU(pcpxflt_GPU(zeta, 2.)), a2);
    ckk = acpx_GPU(c1, c2);

    return( b0*ci*rc / sqrt(rem2)*(ckk.im*x - ckk.re*y) );
}


__device__ type_t module_potential_81_SOA_GPU(const struct point *pImage, const struct Potential_SOA *lens, int shalos, int nhalos){
	//asm volatile("# module_potentialDerivatives_totalGradient_81_SOA begins");
#if 0
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	if(row == 0 && col == 0) printf("# module_potentialDerivatives_totalGradient_81_SOA begins\n");
#endif
	//std::cout << "# module_potentialDerivatives_totalGradient_81_SOA begins" << std::endl;
	// 6 DP loads, i.e. 48 Bytes: position_x, position_y, ellipticity_angle, ellipticity_potential, rcore, b0
	//
	type_t t05;
	type_t potential, pa, ps;
	potential =  0;

for(int i = shalos; i < shalos + nhalos; i++)
{

	struct point true_coord;
	//True coord
	true_coord.x = pImage->x - lens->position_x[i];
	true_coord.y = pImage->y - lens->position_y[i];
	//Rotation
	type_t cose = lens->anglecos[i];
	type_t sine = lens->anglesin[i];
	//
	type_t x = true_coord.x*cose + true_coord.y*sine;
	type_t y = true_coord.y*cose - true_coord.x*sine;
	// 81 comput
	t05 = lens->rcut[i] / (lens->rcut[i] - lens->rcore[i]);

	//if(blockIdx.x*blockDim.x + threadIdx.x == 0 && blockIdx.y*blockDim.y + threadIdx.y == 0)printf("t05 %f rcut: %f, rcore: %f, b0:%f \n",t05, lens->rcut[i],lens->rcore[i], lens->b0[i]);
	pa = pi05_gpu(x, y, lens->ellipticity_potential[i], lens->rcore[i], lens->b0[i]);
	////////////////////////////////////////////////
	ps = pi05_gpu(x, y, lens->ellipticity_potential[i], lens->rcut[i], lens->b0[i]);
	/////////////////////////////////////
	potential += t05 * (pa - ps);
	///////////
}

return(potential);

}

#if 1
typedef type_t (*potential_func_GPU_t) (const struct point *pImage, const struct Potential_SOA *lens, int shalos, int nhalos);

__constant__ potential_func_GPU_t potential_func_GPU[100] =
{
		0, 0, 0, 0, 0, 0, 0, 0, 0,  0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
		0,  module_potential_81_SOA_GPU, 0, 0, 0, 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};
#endif


__global__
void
module_potential_totalPotential_SOA_GPU(type_t *potential_GPU, const struct Potential_SOA *lens, const struct grid_param *frame, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
	struct point image_point;
	type_t potential, potential_temp;
	//
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	//if(row == 0 && col == 0) printf("Start GPU \n");

	//
	if ((row + jstart < nbgridcells_y) && (col + istart < nbgridcells_x))
	{
		int index = row*nbgridcells_x + col;
		// Create temp pot variable to minimise writing to global memory potential
		potential = 0;
		potential_temp = 0;
		//
		image_point.x = frame->xmin + (col + istart)*dx;
		image_point.y = frame->ymin + (row + jstart)*dy;
		//
		int shalos = 0;
		//if(row == 0 && col == 0) printf("Start 2 GPU \n");
		//if(row == 0 && col == 0) std::cout << std::endl;;


		while (shalos < nhalos)
		{
			int lens_type = lens->type[shalos];
			int count     = 1;
			while (lens->type[shalos + count] == lens_type and shalos + count < nhalos) count++;
			//
			//clumpgrad = (*halo_func2_GPU[lens_type])(&image_point, lens, shalos, count);
			//clumpgrad = module_potentialDerivatives_totalGradient2_81_SOA_GPU(&image_point, lens, shalos, count);

			if(lens_type == 81) potential_temp = module_potential_81_SOA_GPU(&image_point, lens, shalos, count);
			//else if(lens_type == 14) clumpgrad = module_potentialDerivatives_totalGradient2_14_SOA_GPU(&image_point, lens, shalos, count);
			else if(row == 0 && col == 0) printf("No kernel selected \n");

			//
			potential += potential_temp;
			shalos += count;
		}
		//if(row == 0 && col == 0) printf(" %f %f %f %f \n",grad.a,grad.b,grad.c,grad.d);
		// Write to global memory
		potential_GPU[index] = potential;
		//if(row == 0 && col == 0) printf("point = %lf \n", grid_grad2_a[index] );

	}
}

