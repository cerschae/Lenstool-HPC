#include "hip/hip_runtime.h"
//
//
//
#include <fstream>
#include "grid_map_GPU.cuh"
#include "gradient2_GPU.cuh"
#include <structure_hpc.hpp>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0


#define cudasafe 

extern "C" 
{
	type_t myseconds();
}
//
//
//void amplif_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t z,int mode_amp,int nhalos,int nbgridcells_x, int nbgridcells_y);
//map_gpu_function_t amplif_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t z,int mode_amp,int nhalos,int nbgridcells_x, int nbgridcells_y);

__global__ void amplif_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t z,int nbgridcells);
//
void map_grid_GPU(map_gpu_function_t mapfunction, type_t *map, const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells,int mode_amp, type_t z )
{
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells - 1);
    type_t dy = (frame->ymax - frame->ymin)/(nbgridcells - 1);
        //
    map_grid_GPU(mapfunction,map,cosmo, frame, lens, nhalos,mode_amp,z, dx, dy, nbgridcells, nbgridcells, 0, 0);
}
//
void map_grid_GPU(map_gpu_function_t mapfunction, type_t *map,const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int mode_amp, type_t z, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{

	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;

	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

	if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
	{
		fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
		exit(-1);
	}
	else
	{
		nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
		// per Block that the GPU supports
	}

	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	type_t *grid_grad2_a_gpu, *grid_grad2_b_gpu , *grid_grad2_c_gpu, *grid_grad2_d_gpu, *map_gpu;

	type_t dl0s = module_cosmodistances_objectObject(lens->z[0], z, *cosmo);
	type_t dos = module_cosmodistances_observerObject(z, *cosmo);

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradient2gpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradient2gpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradient2gpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_a_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_a_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_b_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_b_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_c_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_c_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_d_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_d_gpu: " );
	cudasafe(hipMalloc( (void**)&(map_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc map: " );
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2pu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy fame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	//
	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
	//
	type_t time = -myseconds();
	//
	module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	mapfunction(map_gpu,grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu,dl0s,z,mode_amp,nhalos,nbgridcells_x,nbgridcells_y);
	//amplif_grid_CPU_GPU(map_gpu,grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu,dl0s,z,mode_amp,nhalos,nbgridcells_x,nbgridcells_y);
	//cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	time += myseconds();
	//std::cout << "	kernel time = " << time << " s." << std::endl;
	//


	cudasafe(hipMemcpy( map, map_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_a_gpu: " );
	//cudasafe(hipMemcpy( grid_grad2_b, grid_grad2_b_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost)," --- Gradient2gpu.cu : Copy source_b_gpu: " );
	//cudasafe(hipMemcpy( grid_grad2_c, grid_grad2_c_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_c_gpu: " );
	//cudasafe(hipMemcpy( grid_grad2_d, grid_grad2_d_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost)," --- Gradient2gpu.cu : Copy source_d_gpu: " );
	//
	//printf("-----> %f %f \n",grid_grad_x[Nx], grid_grad_y[Ny]);
	// Free GPU memory
	hipFree(lens_gpu);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(grid_grad2_a_gpu);
	hipFree(grid_grad2_b_gpu);
	hipFree(grid_grad2_c_gpu);
	hipFree(grid_grad2_d_gpu);
	hipFree(map_gpu);
}

void amplif5_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t z,int mode_amp, int nhalos,int nbgridcells_x, int nbgridcells_y)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        int count = nhalos;
        //printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (double) (8*nhalos + BLOCK_SIZE_X*nbgridcells/BLOCK_SIZE_Y)*sizeof(double));
        printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        //module_potentialDerivatives_totalGradient_SOA_GPU<<<grid, threads>>> (grid_grad_x, grid_grad_y, lens, frame, nhalos, nbgridcells_x);
        amplif_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}

__global__
void
amplif_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t z,int nbgridcells)
{
    struct point image_point;
	struct matrix grad, clumpgrad;
	//
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        type_t kappa = (grid_grad2_a[index] + grid_grad2_c[index]) / 2.;
        ampli[index] = kappa;
    }

}

#if 0
void module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nhalos, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        int count = nhalos;
        //printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (double) (8*nhalos + BLOCK_SIZE_X*nbgridcells/BLOCK_SIZE_Y)*sizeof(double));
        printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(grid_grad2_a, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_b, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_c, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        hipMemset(grid_grad2_d, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        //module_potentialDerivatives_totalGradient_SOA_GPU<<<grid, threads>>> (grid_grad_x, grid_grad_y, lens, frame, nhalos, nbgridcells_x);
        module_potentialDerivatives_totalGradient2_SOA_GPU<<<grid, threads>>> (grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,  lens_gpu, frame, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
        cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU_8_SOA_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
//
//
void
module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(type_t *grid_grad2_a, type_t *grid_grad2_b, type_t *grid_grad2_c, type_t *grid_grad2_d, const struct grid_param *frame, const struct Potential_SOA *lens_gpu, int nbgridcells, int nhalos)
{
	//
	int GRID_SIZE_X = (nbgridcells + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
	int GRID_SIZE_Y = (nbgridcells + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y; 
	//
	type_t* timer = (type_t *) malloc((int) nbgridcells*nbgridcells*sizeof(type_t));
	type_t* dtimer;
	//
	dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
	dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);	
	//
	int count = nhalos;	
	//printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*nbgridcells/BLOCK_SIZE_Y)*sizeof(type_t));
	printf("nhalos = %d, size of shared memory = %lf\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
	//
	hipMemset(grid_grad2_a, 0, nbgridcells*nbgridcells*sizeof(type_t));
	hipMemset(grid_grad2_b, 0, nbgridcells*nbgridcells*sizeof(type_t));
	hipMemset(grid_grad2_c, 0, nbgridcells*nbgridcells*sizeof(type_t));
	hipMemset(grid_grad2_d, 0, nbgridcells*nbgridcells*sizeof(type_t));
	//
	//module_potentialDerivatives_totalGradient2_SOA_GPU<<<grid, threads>>> (grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d, lens_gpu, frame, nbgridcells, nhalos);
	cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU_8_SOA_GPU");
	//
	hipDeviceSynchronize();
	printf("GPU kernel done...\n");
}
#endif

