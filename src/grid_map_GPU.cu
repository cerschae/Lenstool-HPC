#include "hip/hip_runtime.h"
/**
* @Author Christoph Schaefer, EPFL (christophernstrerne.schaefer@epfl.ch), Gilles Fourestey (gilles.fourestey@epfl.ch)
* @date   July 2017
* @version 0,1
*
*/

#include <fstream>
#include "grid_map_GPU.cuh"
#include "gradient2_GPU.cuh"
#include <structure_hpc.hpp>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 16

//#define ROT

#define _SHARED_MEM

#ifdef _SHARED_MEM
#define SHARED __shared__
#warning "shared memory"
extern __shared__ type_t shared[];
#else
#define SHARED 
#endif

#define Nx 1
#define Ny 0


#define cudasafe 

extern "C" 
{
	type_t myseconds();
}

__device__ struct  ellipse formeli_HPC(type_t a, type_t b, type_t c);


//GPU mapping function declaration to change when figured out linkage problems
__global__ void amplif_1_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t dos, type_t z,int nbgridcells);
__global__ void amplif_2_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t dos, type_t z,int nbgridcells);
__global__ void amplif_3_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t dos, type_t z,int nbgridcells);
__global__ void amplif_4_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t dos, type_t z,int nbgridcells);
__global__ void amplif_5_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t dos, type_t z,int nbgridcells);
__global__ void amplif_6_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t dos, type_t z,int nbgridcells);
__global__ void mass_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t mult, type_t ds, type_t dl, type_t h, type_t z,int nbgridcells);


////Map function selection
map_gpu_function_t select_map_function(std::string mode, const struct runmode_param* runmode){
	if (mode == "ampli"){
		if(runmode->amplif == 1){
			return &amplif_1_grid_CPU_GPU;
		}
		else if(runmode->amplif == 2){
			return &amplif_2_grid_CPU_GPU;
		}
		else if(runmode->amplif == 3){
			return &amplif_3_grid_CPU_GPU;
		}
		else if(runmode->amplif == 4){
			return &amplif_4_grid_CPU_GPU;
		}
		else if(runmode->amplif == 5){
			return &amplif_5_grid_CPU_GPU;
		}
		else if(runmode->amplif == 6){
			return &amplif_6_grid_CPU_GPU;
		}
		else{
			fprintf(stderr, "ERROR: Amplif mode %d not supported yet \n",runmode->amplif);
			exit(-1);
		}
	}
	else if(mode == "mass"){
		if(runmode->mass == 1){
			return &mass_1_grid_CPU_GPU;
		}
		else if(runmode->mass == 2){
			return &mass_2_grid_CPU_GPU;
		}
		else if(runmode->mass == 3){
			return &mass_3_grid_CPU_GPU;
		}
		else if(runmode->mass == 4){
			return &mass_4_grid_CPU_GPU;
		}
		else{
			fprintf(stderr, "ERROR: Mass mode %d not supported yet \n",runmode->mass);
			exit(-1);
		}
	}
	else{
		fprintf(stderr, "ERROR: No mode recognised \n");
		exit(-1);
	}
	return 0;
}

////Mass Map calculation, doesnt fit the bloody template...
void map_mass_grid_GPU(map_gpu_function_t mapfunction, type_t *map, const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells,int mode_amp, type_t zl, type_t zs )
{
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells - 1);
    type_t dy = (frame->ymax - frame->ymin)/(nbgridcells - 1);
    //
    map_mass_grid_GPU(mapfunction,map,cosmo, frame, lens, nhalos,mode_amp,zl, zs, dx, dy, nbgridcells, nbgridcells, 0, 0);
}
//
void map_mass_grid_GPU(map_gpu_function_t mapfunction, type_t *map,const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int mode_amp,  type_t zl, type_t zs, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{

	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;
	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

	if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
	{
		fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
		exit(-1);
	}
	else
	{
		nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
		// per Block that the GPU supports
	}

	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu, *dlsds_gpu;
	type_t *grid_grad2_a_gpu, *grid_grad2_b_gpu , *grid_grad2_c_gpu, *grid_grad2_d_gpu, *map_gpu;


	if(zl == 0) {
		zl = lens->z[0];
	}

	type_t dl0s = module_cosmodistances_objectObject(zl, zs, *cosmo);
	type_t dos = module_cosmodistances_observerObject(zs, *cosmo);
	type_t dol = module_cosmodistances_observerObject(zl, *cosmo);
	//select_ratio_function(std::string mode, const struct runmode_param* runmode, type_t dls, type_t ds);

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradient2gpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradient2gpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(dlsds_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc dlsds_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradient2gpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_a_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_a_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_b_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_b_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_c_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_c_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_d_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_d_gpu: " );
	cudasafe(hipMalloc( (void**)&(map_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc map: " );
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2pu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(dlsds_gpu, lens->dlsds, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy dlsds: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy fame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	lens_gpu->dlsds 		= dlsds_gpu;
	//
	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
	//
	type_t time = -myseconds();
	//
#if 0
	int shalos = 0;
    while (shalos < nhalos)
    {
            int lens_type = lens->type[shalos];
            type_t z = lens->z[shalos];
            int count     = 1;
            if (shalos + count < nhalos){
            	std::cerr << shalos  << " " <<  count << " " << lens->type[shalos + count] << " " << lens->z[shalos + count] << " " << std::endl;
            	while (lens->type[shalos + count] == lens_type and lens->z[shalos + count] == z ){
            		count++;
            		if(shalos + count >= nhalos)
            			break;
            		//std::cerr << shalos  << " " <<  count << " " << lens->type[shalos + count] << " " << lens->z[shalos + count] << " " << std::endl;
            	}
            	std::cerr << shalos  << " " <<  count << " " << lens->type[shalos + count] << " " << lens->z[shalos + count] << " " << std::endl;
            }
            //if (shalos < nhalos) std::cerr << shalos  << " " <<  count << " " << lens->type[shalos + count] << " " << lens->z[shalos + count] << " " << std::endl;

			shalos += count;
		}
#endif
	//module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	module_potentialDerivatives_Kmap_SOA_CPU_GPU(grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	//std::cerr << "ZS " << zs << " "<< dl0s << " "<< dos <<std::endl;
	mapfunction(map_gpu,grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu,dl0s,dos,dol,cosmo->h,zs,nbgridcells_x,nbgridcells_y,frame);
	cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	//
	cudasafe(hipMemcpy( map, map_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_a_gpu: " );
	//
	time += myseconds();
	std::cout << "	kernel time = " << time << " s." << std::endl;
	//printf("-----> %f %f \n",grid_grad_x[Nx], grid_grad_y[Ny]);
	// Free GPU memory
	hipFree(lens_gpu);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(dlsds_gpu);
	hipFree(grid_grad2_a_gpu);
	hipFree(grid_grad2_b_gpu);
	hipFree(grid_grad2_c_gpu);
	hipFree(grid_grad2_d_gpu);
	hipFree(map_gpu);
}

////General Map calculation
void map_grid_GPU(map_gpu_function_t mapfunction, type_t *map, const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos ,int nbgridcells,int mode_amp, type_t z )
{
	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells - 1);
    type_t dy = (frame->ymax - frame->ymin)/(nbgridcells - 1);
    //
    map_grid_GPU(mapfunction,map,cosmo, frame, lens, nhalos,mode_amp,z, dx, dy, nbgridcells, nbgridcells, 0, 0);
}
//
void map_grid_GPU(map_gpu_function_t mapfunction, type_t *map,const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int mode_amp, type_t z, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{

	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;
	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

	if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
	{
		fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
		exit(-1);
	}
	else
	{
		nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
		// per Block that the GPU supports
	}

	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	type_t *grid_grad2_a_gpu, *grid_grad2_b_gpu , *grid_grad2_c_gpu, *grid_grad2_d_gpu, *map_gpu;

	type_t dl0s = module_cosmodistances_objectObject(lens->z[0], z, *cosmo);
	type_t dos = module_cosmodistances_observerObject(z, *cosmo);
	type_t dol = module_cosmodistances_observerObject(lens->z[0], *cosmo);
	//select_ratio_function(std::string mode, const struct runmode_param* runmode, type_t dls, type_t ds)

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradient2gpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradient2gpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradient2gpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_a_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_a_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_b_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_b_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_c_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_c_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_d_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_d_gpu: " );
	cudasafe(hipMalloc( (void**)&(map_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc map: " );
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2pu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy fame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	//
	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
	//
	type_t time = -myseconds();
	//
	module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	mapfunction(map_gpu,grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu,dl0s,dos,dol,cosmo->h,z,nbgridcells_x,nbgridcells_y,frame);
	//cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	//
	cudasafe(hipMemcpy( map, map_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_a_gpu: " );
	//
	time += myseconds();
	std::cout << "	kernel time = " << time << " s." << std::endl;
	// Free GPU memory
	hipFree(lens_gpu);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(grid_grad2_a_gpu);
	hipFree(grid_grad2_b_gpu);
	hipFree(grid_grad2_c_gpu);
	hipFree(grid_grad2_d_gpu);
	hipFree(map_gpu);
}


//allows for resizing of the source
void map_resizedgrid_GPU(map_gpu_function_t mapfunction, type_t *map,const struct cosmo_param *cosmo, const struct grid_param *frame, const struct Potential_SOA *lens, int nhalos, int mode_amp, type_t z, type_t dx, type_t dy, int nbgridcells_x, int nbgridcells_y, int istart, int jstart)
{

	int nBlocks_gpu = 0;
	// Define the number of threads per block the GPU will use
	hipDeviceProp_t properties_gpu;
	hipGetDeviceProperties(&properties_gpu, 0); // Get properties of 0th GPU in use

	if (properties_gpu.maxThreadsDim[0]<threadsPerBlock)
	{
		fprintf(stderr, "ERROR: The GPU has to support at least %u threads per block.\n", threadsPerBlock);
		exit(-1);
	}
	else
	{
		nBlocks_gpu = properties_gpu.maxGridSize[0] / threadsPerBlock;  // Get the maximum number of blocks with the chosen number of threads
		// per Block that the GPU supports
	}

	//Create resized frame information for source amplification and the like
	grid_param resized_frame;
    type_t resized_dx = (frame->xmax - frame->xmin) / 6.;
    type_t resized_dy = (frame->ymax - frame->ymin) / 6.;
	resized_frame.xmax = frame->xmax - resized_dx;
	resized_frame.xmin = frame->xmin + resized_dx;
	resized_frame.ymax = frame->ymax - resized_dy;
	resized_frame.ymin = frame->xmin + resized_dy;


	grid_param *frame_gpu;
	Potential_SOA *lens_gpu,*lens_kernel;
	int *type_gpu;
	type_t *lens_x_gpu, *lens_y_gpu, *b0_gpu, *angle_gpu, *epot_gpu, *rcore_gpu, *rcut_gpu, *anglecos_gpu, *anglesin_gpu;
	type_t *grid_grad2_a_gpu, *grid_grad2_b_gpu , *grid_grad2_c_gpu, *grid_grad2_d_gpu, *map_gpu;

	type_t dl0s = module_cosmodistances_objectObject(lens->z[0], z, *cosmo);
	type_t dos = module_cosmodistances_observerObject(z, *cosmo);
	//select_ratio_function(std::string mode, const struct runmode_param* runmode, type_t dls, type_t ds)

	lens_gpu = (Potential_SOA *) malloc(sizeof(Potential_SOA));
	lens_gpu->type = (int *) malloc(sizeof(int));

	// Allocate variables on the GPU
	cudasafe(hipMalloc( (void**)&(lens_kernel), sizeof(Potential_SOA)),"Gradient2gpu.cu : Alloc Potential_SOA: " );
	cudasafe(hipMalloc( (void**)&(type_gpu), nhalos*sizeof(int)),"Gradient2gpu.cu : Alloc type_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_x_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc x_gpu: " );
	cudasafe(hipMalloc( (void**)&(lens_y_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc y_gpu: " );
	cudasafe(hipMalloc( (void**)&(b0_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc b0_gpu: " );
	cudasafe(hipMalloc( (void**)&(angle_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc angle_gpu: " );
	cudasafe(hipMalloc( (void**)&(epot_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc epot_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcore_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcore_gpu: " );
	cudasafe(hipMalloc( (void**)&(rcut_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc rcut_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglecos_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglecos_gpu: " );
	cudasafe(hipMalloc( (void**)&(anglesin_gpu), nhalos*sizeof(type_t)),"Gradient2gpu.cu : Alloc anglesin_gpu: " );
	cudasafe(hipMalloc( (void**)&(frame_gpu), sizeof(grid_param)),"Gradient2gpu.cu : Alloc frame_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_a_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_a_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_b_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_b_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_c_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_c_gpu: " );
	cudasafe(hipMalloc( (void**)&(grid_grad2_d_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc source_d_gpu: " );
	cudasafe(hipMalloc( (void**)&(map_gpu), (nbgridcells_x) * (nbgridcells_y) *sizeof(type_t)),"Gradient2gpu.cu : Alloc map: " );
	// Copy values to the GPU
	//
	cudasafe(hipMemcpy(type_gpu,lens->type , nhalos*sizeof(int),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy type_gpu: " );
	cudasafe(hipMemcpy(lens_x_gpu,lens->position_x , nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy x_gpu: " );
	cudasafe(hipMemcpy(lens_y_gpu,lens->position_y , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy y_gpu: " );
	cudasafe(hipMemcpy(b0_gpu,lens->b0 , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2pu.cu : Copy b0_gpu: " );
	cudasafe(hipMemcpy(angle_gpu,lens->ellipticity_angle , nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy angle_gpu: " );
	cudasafe(hipMemcpy(epot_gpu, lens->ellipticity_potential, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy epot_gpu: " );
	cudasafe(hipMemcpy(rcore_gpu, lens->rcore, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy rcore_gpu: " );
	cudasafe(hipMemcpy(rcut_gpu, lens->rcut, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy rcut_gpu: " );
	cudasafe(hipMemcpy(anglecos_gpu, lens->anglecos, nhalos*sizeof(type_t),hipMemcpyHostToDevice ),"Gradient2gpu.cu : Copy anglecos: " );
	cudasafe(hipMemcpy(anglesin_gpu, lens->anglesin, nhalos*sizeof(type_t), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy anglesin: " );
	cudasafe(hipMemcpy(frame_gpu, frame, sizeof(grid_param), hipMemcpyHostToDevice),"Gradient2gpu.cu : Copy fame_gpu: " );
	//
	lens_gpu->type 			= type_gpu;
	lens_gpu->position_x 		= lens_x_gpu;
	lens_gpu->position_y 		= lens_y_gpu;
	lens_gpu->b0 			= b0_gpu;
	lens_gpu->ellipticity_angle 	= angle_gpu;
	lens_gpu->ellipticity_potential = epot_gpu;
	lens_gpu->rcore 		= rcore_gpu;
	lens_gpu->rcut 			= rcut_gpu;
	lens_gpu->anglecos 		= anglecos_gpu;
	lens_gpu->anglesin 		= anglesin_gpu;
	//
	hipMemcpy(lens_kernel, lens_gpu, sizeof(Potential_SOA), hipMemcpyHostToDevice);
	//
	type_t time = -myseconds();
	//
	module_potentialDerivatives_totalGradient2_SOA_CPU_GPU(grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu, frame_gpu, lens_kernel, nhalos, dx, dy, nbgridcells_x, nbgridcells_y, istart, jstart);
	//
	//mapfunction(map_gpu,grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu,dl0s,dos,z,mode_amp,nhalos,nbgridcells_x,nbgridcells_y);
	//amplif_grid_CPU_GPU(map_gpu,grid_grad2_a_gpu, grid_grad2_b_gpu, grid_grad2_c_gpu, grid_grad2_d_gpu,dl0s,z,mode_amp,nhalos,nbgridcells_x,nbgridcells_y);
	//cudasafe(hipGetLastError(), "module_potentialDerivative_totalGradient_SOA_CPU_GPU");
	hipDeviceSynchronize();
	//
	cudasafe(hipMemcpy( map, map_gpu, (nbgridcells_x)*(nbgridcells_y)*sizeof(type_t), hipMemcpyDeviceToHost )," --- Gradient2gpu.cu : Copy source_a_gpu: " );
	//
	time += myseconds();
	std::cout << "	kernel time = " << time << " s." << std::endl;
	//printf("-----> %f %f \n",grid_grad_x[Nx], grid_grad_y[Ny]);
	// Free GPU memory
	hipFree(lens_gpu);
	hipFree(type_gpu);
	hipFree(lens_x_gpu);
	hipFree(lens_y_gpu);
	hipFree(b0_gpu);
	hipFree(angle_gpu);
	hipFree(epot_gpu);
	hipFree(rcore_gpu);
	hipFree(rcut_gpu);
	hipFree(anglecos_gpu);
	hipFree(anglesin_gpu);
	hipFree(grid_grad2_a_gpu);
	hipFree(grid_grad2_b_gpu);
	hipFree(grid_grad2_c_gpu);
	hipFree(grid_grad2_d_gpu);
	hipFree(map_gpu);
}


////Map functions
//Amplification NR 1
void amplif_1_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        amplif_1_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s,ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void amplif_1_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	type_t A,B,C;
	ellipse amp;
	type_t dlsds= dl0s/ds;
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        A = 1. - grid_grad2_a[index]*dlsds;   // 1 - DLS/DS * d2phixx
        B = - grid_grad2_b[index]*dlsds;   // - DLS/DS * d2phixy
        C = 1. - grid_grad2_c[index]*dlsds;   // 1 - DLS/DS * d2phiyy
        amp = formeli_HPC(A, B, C);
        ampli[index] = 1. / (amp.a * amp.b);
    }
}
//Amplification NR 2
void amplif_2_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        amplif_2_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s,ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void amplif_2_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	type_t A,B,C;
	ellipse amp;
	type_t dlsds= dl0s/ds;
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        A = 1. - grid_grad2_a[index]*dlsds;   // 1 - DLS/DS * d2phixx
        B = - grid_grad2_b[index]*dlsds;   // - DLS/DS * d2phixy
        C = 1. - grid_grad2_c[index]*dlsds;   // 1 - DLS/DS * d2phiyy
        amp = formeli_HPC(A, B, C);
        ampli[index] = 1. / fabs(amp.a * amp.b);
    }
}
//Amplification NR 3
void amplif_3_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        amplif_3_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s,ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void amplif_3_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	type_t A,B,C;
	ellipse amp;
	type_t dlsds= dl0s/ds;
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        A = 1. - grid_grad2_a[index]*dlsds;   // 1 - DLS/DS * d2phixx
        B = - grid_grad2_b[index]*dlsds;   // - DLS/DS * d2phixy
        C = 1. - grid_grad2_c[index]*dlsds;   // 1 - DLS/DS * d2phiyy
        amp = formeli_HPC(A, B, C);
        ampli[index] = -2.5 * log10(fabs(amp.a * amp.b));
    }
}
//Amplification NR 4
void amplif_4_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        amplif_4_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s,ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void amplif_4_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	//type_t kappa,ga1,ga2,gam,gp;
	////
	type_t dlsds = dl0s/ds;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {

    	int index = row*nbgridcells + col;
        type_t kappa = (grid_grad2_a[index] + grid_grad2_c[index])*dlsds / 2.;
        type_t ga1 = (grid_grad2_a[index] - grid_grad2_c[index])*dlsds / 2.;
        type_t ga2 = grid_grad2_b[index]*dlsds;
        type_t gam = sqrt(ga1 * ga1 + ga2 * ga2);
        type_t gp = gam / (1 - kappa);

        ampli[index] = (1 - kappa) * (1 + gp * gp) / (1 - gp * gp);
    }
}
//Amplification NR 5
void amplif_5_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        amplif_5_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s,ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void amplif_5_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        type_t kappa = (grid_grad2_a[index] + grid_grad2_c[index]) / 2.;
        ampli[index] = kappa;
    }
}
//Amplification NR 6
void amplif_6_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        amplif_6_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dl0s, ds,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void amplif_6_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t z,int nbgridcells)
{
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        type_t ga1 = (grid_grad2_a[index] - grid_grad2_c[index]) / 2.;
        type_t ga2 = grid_grad2_b[index];
        type_t gam = sqrt(ga1 * ga1 + ga2 * ga2);
        ampli[index] = gam;
    }
}

//Mass NR 1
void mass_1_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        mass_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,1, dl0s,ds,h,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//Mass NR 2
void mass_2_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        type_t dcrit = cH4piG * h / dl / dl0s * ds;  // in g/cm^2
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        mass_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dcrit, dl0s,ds,h,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//Mass NR 3
void mass_3_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
    	type_t dx = (frame->xmax - frame->xmin)/(nbgridcells_x - 1);
        type_t dy = (frame->ymax - frame->ymin)/(nbgridcells_y - 1);
        //printf("grid_size_x = %d, grid_size_y = %d, nbgridcells_x = %d, nbgridcells_y = %d, istart = %d, jstart = %d (split)\n", GRID_SIZE_X, GRID_SIZE_Y, nbgridcells_x, nbgridcells_y, istart, jstart);
        type_t conv = MCRIT12 / h * dx * dy * dl / dl0s * ds;  // in  10^12 M_sol/pixel
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        mass_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,conv, dl0s,ds,h,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//Mass NR 4
void mass_4_grid_CPU_GPU(type_t *map,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t dl0s, type_t ds, type_t dl, type_t h, type_t z, int nbgridcells_x, int nbgridcells_y, const struct grid_param *frame)
{
        int GRID_SIZE_X = (nbgridcells_x + BLOCK_SIZE_X - 1)/BLOCK_SIZE_X; // number of blocks
        int GRID_SIZE_Y = (nbgridcells_y + BLOCK_SIZE_Y - 1)/BLOCK_SIZE_Y;
        //
        type_t dcritA = cH0_4piG * h / dl / dl0s * ds;  // in 10^12 M_sol/kpc^2
        //
        dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y/1);
        dim3 grid   (GRID_SIZE_X , GRID_SIZE_Y);
        //
        //printf("nhalos = %d, size of shared memory = %lf (split)\n", nhalos, (type_t) (8*nhalos + BLOCK_SIZE_X*BLOCK_SIZE_Y)*sizeof(type_t));
        //
        hipMemset(map, 0, nbgridcells_x*nbgridcells_y*sizeof(type_t));
        //
        mass_grid_GPU<<<grid, threads>>> (map,grid_grad2_a, grid_grad2_b,grid_grad2_c, grid_grad2_d,dcritA, dl0s,ds,h,z,nbgridcells_x);
        cudasafe(hipGetLastError(), "amplif_grid_CPU_GPU");
        //
        hipDeviceSynchronize();
        printf("GPU kernel done...\n");
}
//
__global__ void mass_grid_GPU(type_t *ampli,type_t *grid_grad2_a,type_t *grid_grad2_b,type_t *grid_grad2_c,type_t *grid_grad2_d, type_t mult, type_t dls, type_t ds,type_t h, type_t z,int nbgridcells)
{
	////
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    //type_t dlsds = dls / ds ;
    //
    if ((row < nbgridcells) && (col < nbgridcells))
    {
    	int index = row*nbgridcells + col;
        type_t ga1 = (grid_grad2_a[index] + grid_grad2_c[index]) * 0.5 ;//* dlsds ;
        ampli[index] = ga1*mult;
    }
}

__device__ struct  ellipse formeli_HPC(type_t a, type_t b, type_t c)
{
    struct  ellipse eli;
    type_t  e, delta, lambda, mu;

    // eq carateristique : det(M-xI) = 0
    delta = (a - c)*(a - c) + 4*b*b;    //  4*gamma^2 (cf phd_JPK eq 2.56)
    e = sqrt(delta);  /*e is 2 * shear, ie 2*gamma*/
    lambda = .5*(a + c + e);  // 1 - k + gamma
    mu = .5*(a + c - e);      // 1 - k - gamma

    eli.a = lambda;
    eli.b = mu;
    if (lambda != mu && fabs(b) > 1e-5)
        eli.theta = atan2(lambda - a, b); // cf phd_JPK eq 2.58, and
    // tan(theta)= ( -cos(2theta) +- 1 ) / sin(2theta)
// ADDED by EJ 29/11/2007
    else if ( a >= c ) // ellipse aligned along the major axis of magnification
        eli.theta = 0.;
    else
        eli.theta = acos(-1.) / 2.;    // ellipse aligned along the minor axis of magnification

    return(eli);
}

